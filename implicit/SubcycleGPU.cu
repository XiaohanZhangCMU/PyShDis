
#include <hip/hip_runtime.h>
/*-------------------------------------------------------------------------
 *
 *	This file contains the GPU implementation of the subcycling
 * 	time-integrator algorithm.
 * 
 * 	As of now, the GPU implementation has the following limitations:
 * 		- 	it can only be run with 1 CPU (serial mode)
 * 		- 	it can only be used with the FMM and cannot be used 
 * 			with the Rijm table
 * 		-	it can only be used with the FCC_0 mobility law
 * 		- 	it can not be used with rotated frames (must set useLabFrame = 0)
 * 		-	it cannot be used with interactions in subgroup 1
 * 			(i.e. one must set rg1 = 0)
 * 
 * 	Nicolas Bertin, 06/27/2017
 *
 *-----------------------------------------------------------------------*/

#ifdef _GPU_SUBCYCLE

#include <iostream>
#include <fstream>
#include <stdio.h>
#include <cmath>
#include <cstdlib>
#include <sys/time.h>
#include <vector>
#include <algorithm>
#include <cuda.h>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>

#include "Util.h"
#include "Home.h"
#include "SubcycleGPU.h"
#include "Mobility.h"

/*------------------------------------------------------------------------
 *
 *      Function:    HandleErrorGPU
 *
 *-----------------------------------------------------------------------*/
#define HANDLE_ERROR(err) (HandleErrorGPU(err, __FILE__, __LINE__ ))
static void HandleErrorGPU(cudaError_t err, const char *file, int line) {
	if (err != cudaSuccess) {
		printf( "%s in %s at line %d\n", cudaGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}

/*------------------------------------------------------------------------
 *
 *      Function:    CheckErrorGPU
 *
 *-----------------------------------------------------------------------*/
void CheckErrorGPU(const char *message) {
	cudaError_t error = cudaGetLastError();
	if(error != cudaSuccess) {
		fprintf(stderr,"ERROR %s: %s\n", message, cudaGetErrorString(error));
		exit(-1);
	}                        
}

/*------------------------------------------------------------------------
 *
 *      Function:    SetVariablesGPU
 *
 *-----------------------------------------------------------------------*/
__device__ double  MU, NU, a;
__device__ double  TensionFactor;
__device__ double  MobEdge, MobScrew;
__device__ double  Lx, Ly, Lz;
__device__ double  invLx, invLy, invLz;
__device__ double  rTol, rTolth, rTolrel;
__device__ double  coreEnergy;
__device__ double3 boxc;

__global__ void SetVariablesGPU(double _MU, double _NU, double _a, double _tf, 
                                double _MobEdge, double _MobScrew,
                                double _Lx, double _Ly, double _Lz,
                                double _xc, double _yc, double _zc,
                                double _rTol, double _rTolth, double _rTolrel, double _Ecore)
{
	MU = _MU;
	NU = _NU;
	a = _a;
	TensionFactor = _tf;
	MobEdge = _MobEdge;
	MobScrew = _MobScrew;
	Lx = _Lx;
	Ly = _Ly;
	Lz = _Lz;
	if (_Lx == 0.0) invLx = 0.0;
	else invLx = 1.0 / _Lx;
	if (_Ly == 0.0) invLy = 0.0;
	else invLy = 1.0 / _Ly;
	if (_Lz == 0.0) invLz = 0.0;
	else invLz = 1.0 / _Lz;
	boxc.x = _xc;
	boxc.y = _yc;
	boxc.z = _zc;
	rTol = _rTol;
	rTolth = _rTolth;
	rTolrel = _rTolrel;
	coreEnergy = _Ecore;
}

__device__ void SegSegForceIsotropicGPU(double3 r1, double3 r2, double3 r3, double3 r4, double3 b1, double3 b2,
                                        double3 &fn1, double3 &fn2, double3 &fn3, double3 &fn4);
__device__ void SegSegForceIsotropicCorrGPU(double3 r1, double3 r2, double3 r3, double3 r4, double3 b1, double3 b2,
                                            double3 &fn1, double3 &fn2, double3 &fn3, double3 &fn4);
__device__ void SpecialSegSegForceHalfGPU(double3 r1, double3 r2, double3 r3, double3 r4, 
                                          double3 b1, double3 b2, double3 &fn3, double3 &fn4);

/*------------------------------------------------------------------------
 *
 *      Function:    SpecialSegSegForceHalfGPU
 *
 *-----------------------------------------------------------------------*/
__device__ void SpecialSegSegForceHalfGPU(double3 r1, double3 r2, double3 r3, double3 r4, 
                                          double3 b1, double3 b2, double3 &fn3, double3 &fn4)
{
        int i, j , alt1[3]={1,2,0}, alt2[3]={2,0,1};
        real8 eps, ecrit, c, a2, d2, a2_d2, a2d2inv;
        real8 x1[3], x2[3], x3[3], x4[3], b[3], bp[3];
        real8 f3[3], f4[3];
        real8 vec1[3], vec2[3], t[3], nd[3];
        real8 temp1;
        real8 R[3], Rdt, x1mod[3], x2mod[3];
        real8 oneoverL;
        real8 y[2], z[2], yv[4], zv[4], ypz[4], ymz[4];
        real8 Ra[4], Rainv[4], Log_Ra_ypz[4];
        real8 temp, tmp[8];
        real8 common1[4], common2[3], common3[3];
        real8 magdiff, diffMag2, x1modMag2, x2modMag2;
        real8 f_003v[4], f_103v[4], f_113v[4], f_213v[4];
        real8 f_005v[4], f_105v[4], f_115v[4], f_215v[4];
        real8 f_003, f_103, f_113, f_213;
        real8 f_005, f_105, f_115, f_215;
        real8 Fint_003, Fint_113, Fint_005, Fint_115;
        real8 I_003[3], I_113[3], I_005[3], I_115[3];
        real8 m4p, m8p, m4pn, a2m4pn, a2m8p;
        real8 tdb, tdbp, nddb, bpctdb, bpctdnd;
        real8 bct[3], bpct[3], ndct[3], bpctct[3];
        real8 cotanthetac;
        real8 pivalue=3.141592653589793;

		ecrit  = 1e-4;
        cotanthetac = sqrt((1 - ecrit*1.01) / (ecrit*1.01));
        
        eps    = 1e-12;
        a2     = a*a;
        m4p    = 0.25 * MU / pivalue;
        m8p    = 0.5 * m4p;
        m4pn   = m4p / ( 1 - NU );
        a2m4pn = a2 * m4pn;
        a2m8p  = a2 * m8p;
            
        fn3.x = 0.0;
        fn3.y = 0.0;
        fn3.z = 0.0;
         
        fn4.x = 0.0;
        fn4.y = 0.0;
        fn4.z = 0.0;
        
        x1[0]=r1.x;
        x1[1]=r1.y;
        x1[2]=r1.z;
        x2[0]=r2.x;
        x2[1]=r2.y;
        x2[2]=r2.z;
        x3[0]=r3.x;
        x3[1]=r3.y;
        x3[2]=r3.z;
        x4[0]=r4.x;
        x4[1]=r4.y;
        x4[2]=r4.z;
        
        b[0]=b2.x;
        b[1]=b2.y;
        b[2]=b2.z;
        bp[0]=b1.x;
        bp[1]=b1.y;
        bp[2]=b1.z;
        
        #pragma unroll
        for(i=0;i<3;i++) { 
            vec1[i]=x4[i]-x3[i];
            vec2[i]=x2[i]-x1[i];
        }

        temp1=0.0e0;
        
        #pragma unroll
        for(i=0;i<3;i++) { 
            temp1+=vec1[i]*vec1[i];
        }

        oneoverL =1/sqrt(temp1);
        
        for(i=0;i<3;i++) { 
            t[i]=vec1[i]*oneoverL;
        }
        
        c=0.0e0;
        
        #pragma unroll
        for(i=0;i<3;i++) { 
            c+=t[i]*vec2[i];
        }

        if (c < 0) {
			#pragma unroll
            for(i=0;i<3;i++) { 
                temp=x2[i];
                x2[i]=x1[i];
                x1[i]=temp;
                bp[i]=-bp[i];
                vec2[i]=-vec2[i];
            }         
        }
/*
 *      Find f3 and f4, but only if at least one of the segment
 *      endpoints is local to the domain.
 */
        temp=0.0e0;
        
        #pragma unroll
        for (i=0;i<3;i++) {
            temp+=vec2[i]*t[i];
        }
        
        #pragma unroll
        for (i=0;i<3;i++) {
            x2mod[i]=x1[i]+temp*t[i];
        }
        
        #pragma unroll
        for (i=0;i<3;i++) {
            vec2[i]=x2[i]-x2mod[i];
        }
               
        temp=0.0e0;
        
        #pragma unroll
        for (i=0;i<3;i++) {
            temp+=vec2[i]*vec2[i];
        }
            
        magdiff=sqrt(temp);
        temp=magdiff*0.5e0 * cotanthetac;
        
        #pragma unroll
        for (i=0;i<3;i++) {
            vec1[i]=temp*t[i];
        }
        
        #pragma unroll
        for (i=0;i<3;i++) {
            x1mod[i]=x1[i]+0.5e0*vec2[i]+vec1[i];
            x2mod[i]+=0.5e0*vec2[i]-vec1[i];
        }
        
        #pragma unroll
        for (i=0;i<3;i++) {
            R[i]=0.5e0*((x3[i]+x4[i])-(x1mod[i]+x2mod[i]));
        }
        
        Rdt=0.0e0;
        
        #pragma unroll
        for (i=0;i<3;i++) {
            Rdt+=R[i]*t[i];
        }
        
        #pragma unroll
        for (i=0;i<3;i++) {
            nd[i]=R[i]-Rdt*t[i];
        }
        
        d2=0.0e0;
        
        #pragma unroll
        for (i=0;i<3;i++) {
            d2+=nd[i]*nd[i];
        }    
        
        #pragma unroll
        for (j=0;j<2;j++) {
            y[j]=0.0e0;
            z[j]=0.0e0;
        }  
        
        #pragma unroll
        for (i=0;i<3;i++) {
            y[0]+=x3[i]*t[i];
            y[1]+=x4[i]*t[i];
            z[0]+=-x1mod[i]*t[i];
            z[1]+=-x2mod[i]*t[i];
        } 
        
        #pragma unroll
        for (j=0;j<2;j++) {
            yv[2*j]=y[j];
            yv[2*j+1]=y[j];
            zv[j]=z[j];
            zv[j+2]=z[j];
        }    
            
        a2_d2 = a2 + d2;   
        
        #pragma unroll
        for (j=0;j<4;j++) {
            ypz[j] = yv[j] + zv[j];
            ymz[j] = yv[j] - zv[j];
        }
        
        #pragma unroll
        for (j=0;j<4;j++) {
            tmp[j]=a2_d2 + ypz[j]*ypz[j];
        }
        
        #pragma unroll
        for (j=0;j<4;j++) {
            Ra[j]=sqrt(tmp[j]);
        }
        
        #pragma unroll
        for (j=0;j<4;j++) {
            Rainv[j]=1.0e0/Ra[j];
        }

        a2d2inv = 1.0e0 / a2_d2;
        
        #pragma unroll
        for (j=0;j<4;j++) {
            tmp[j]=Ra[j] + ypz[j];
			tmp[j+4]=Ra[j]-ypz[j];
        }
        
        #pragma unroll
        for (j=0;j<4;j++) {
            Log_Ra_ypz[j]=0.5e0*(log(tmp[j])-log(tmp[j+4]));
        }
        
        #pragma unroll
        for (j=0;j<4;j++) {
            common1[j] = ymz[j] * Ra[j] * a2d2inv;
            f_115v[j] = -a2d2inv * ypz[j] * Rainv[j];
        }
        
        temp=2.0e0*a2d2inv;
        
        #pragma unroll
        for (j=0;j<4;j++) {
            f_003v[j] = Ra[j];
            f_103v[j] = Log_Ra_ypz[j] - common1[j];
            f_113v[j] = -Log_Ra_ypz[j];
            f_213v[j] = zv[j]*Log_Ra_ypz[j] - Ra[j];
            f_005v[j] = temp*Ra[j] - Rainv[j];
            f_105v[j] = common1[j] - yv[j]*Rainv[j];
            f_215v[j] =  Rainv[j] - zv[j] * f_115v[j];
        }
        
        f_003 = 0.0e0;
        f_103 = 0.0e0;
        f_113 = 0.0e0;
        f_213 = 0.0e0;
        f_005 = 0.0e0;
        f_105 = 0.0e0;
        f_115 = 0.0e0;
        f_215 = 0.0e0;
        
        #pragma unroll
        for (j=1;j<3;j++) {
            f_003v[j] = -f_003v[j];
            f_103v[j] = -f_103v[j];
            f_113v[j] = -f_113v[j];
            f_213v[j] = -f_213v[j];
            f_005v[j] = -f_005v[j];
            f_105v[j] = -f_105v[j];
            f_115v[j] = -f_115v[j];
            f_215v[j] = -f_215v[j];
        }
        
        #pragma unroll
        for (j=0;j<4;j++) {
            f_003 += f_003v[j];
            f_103 += f_103v[j];
            f_113 += f_113v[j];
            f_213 += f_213v[j];
            f_005 += f_005v[j];
            f_105 += f_105v[j];
            f_115 += f_115v[j];
            f_215 += f_215v[j];
        }

        f_103 *= -0.5e0;    
        f_003 *=  a2d2inv;
        f_005 *=  a2d2inv;
        f_105 *=  a2d2inv;  
          
        #pragma unroll
        for (i=0;i<3;i++) {
            bct[i]=b[alt1[i]]*t[alt2[i]] - b[alt2[i]]*t[alt1[i]];
            bpct[i]=bp[alt1[i]]*t[alt2[i]] - bp[alt2[i]]*t[alt1[i]];
            ndct[i]=nd[alt1[i]]*t[alt2[i]] - nd[alt2[i]]*t[alt1[i]];
        }
        
        tdb=0.0e0;
        tdbp=0.0e0;
        nddb=0.0e0;
        bpctdb=0.0e0;
        bpctdnd=0.0e0;
        
        #pragma unroll
        for (i=0;i<3;i++) {
            tdb += t[i]*b[i];
            tdbp+= t[i]*bp[i];
            nddb+= nd[i]*b[i];
            bpctdb += bpct[i]*b[i];
            bpctdnd += bpct[i]*nd[i];
            
        }
            
        temp = tdb*tdbp;
        
        #pragma unroll
        for (i=0;i<3;i++) {
            bpctct[i] = tdbp*t[i] - bp[i];
            common2[i] = temp*nd[i];
            common3[i] = bpctdnd*bct[i];
        }   

        tmp[0]=(m4pn-m4p)*tdb;
        tmp[1]=m4pn*bpctdnd*nddb;
        tmp[2]=a2m8p*tdb;
        tmp[3]=m4pn*bpctdnd*tdb;
        
        #pragma unroll
        for (i=0;i<3;i++) {
            I_003[i] = m4pn*(nddb*bpctct[i] + bpctdb*ndct[i] - common3[i]) -
                       m4p*common2[i]; 
            I_113[i] =  tmp[0]*bpctct[i];
            I_005[i] = -a2m8p*common2[i] - a2m4pn*common3[i] - tmp[1]*ndct[i];
            I_115[i] = -tmp[2]*bpctct[i] - tmp[3]*ndct[i];
        }
                     
        Fint_003 = f_103 - y[0]*f_003;
        Fint_113 = f_213 - y[0]*f_113;
        Fint_005 = f_105 - y[0]*f_005;
        Fint_115 = f_215 - y[0]*f_115;
        
        #pragma unroll
        for (i=0;i<3;i++) {
            f4[i] = (I_003[i]*Fint_003 + I_113[i]*Fint_113 + I_005[i]*Fint_005 +
                     I_115[i]*Fint_115) * oneoverL;
        }

        Fint_003 = y[1]*f_003 - f_103;
        Fint_113 = y[1]*f_113 - f_213;
        Fint_005 = y[1]*f_005 - f_105;
        Fint_115 = y[1]*f_115 - f_215;
        
        #pragma unroll
        for (i=0;i<3;i++) {
            f3[i] = (I_003[i]*Fint_003 + I_113[i]*Fint_113 + I_005[i]*Fint_005 +
                     I_115[i]*Fint_115) * oneoverL;
        }   
        
        fn3.x = f3[0];
        fn3.y = f3[1];
        fn3.z = f3[2];
        fn4.x = f4[0];
        fn4.y = f4[1];
        fn4.z = f4[2];
        
        x1modMag2 = 0.0e0;
        x2modMag2 = 0.0e0;
        
        #pragma unroll
        for (i=0;i<3;i++) {
            x1modMag2 += x1mod[i]*x1mod[i];
            x2modMag2 += x2mod[i]*x2mod[i];
        }

        diffMag2 = magdiff*magdiff;
        
        if (diffMag2 > (eps * (x1modMag2+x2modMag2))) {
			
			double3 fn3cor, fn4cor, fw, fq, bx1, bx2;
			double3 rx1, rx2, rx3, rx4, rx1mod, rx2mod;
			
			rx1.x = x1[0]; rx1.y = x1[1]; rx1.z = x1[2];
			rx2.x = x2[0]; rx2.y = x2[1]; rx2.z = x2[2];
			rx3.x = x3[0]; rx3.y = x3[1]; rx3.z = x3[2];
			rx4.x = x4[0]; rx4.y = x4[1]; rx4.z = x4[2];
			
			rx1mod.x = x1mod[0]; rx1mod.y = x1mod[1]; rx1mod.z = x1mod[2];
			rx2mod.x = x2mod[0]; rx2mod.y = x2mod[1]; rx2mod.z = x2mod[2];
			
			bx1.x = bp[0]; bx1.y = bp[1]; bx1.z = bp[2];
			bx2.x = b[0]; bx2.y = b[1]; bx2.z = b[2];
			
            SegSegForceIsotropicCorrGPU(rx1, rx1mod, rx3, rx4, bx1, bx2, fw, fq, fn3cor, fn4cor);
            
            fn3.x += fn3cor.x;
            fn3.y += fn3cor.y;
            fn3.z += fn3cor.z;
            fn4.x += fn4cor.x;
            fn4.y += fn4cor.y;
            fn4.z += fn4cor.z;
            
            SegSegForceIsotropicCorrGPU(rx2mod, rx2, rx3, rx4, bx1, bx2, fw, fq, fn3cor, fn4cor);
                       
            fn3.x += fn3cor.x;
            fn3.y += fn3cor.y;
            fn3.z += fn3cor.z;
            fn4.x += fn4cor.x;
            fn4.y += fn4cor.y;
            fn4.z += fn4cor.z;
        }
        
        return;
}

/*------------------------------------------------------------------------
 *
 *      Function:    SegSegForceIsotropicCorrGPU
 *
 *-----------------------------------------------------------------------*/
__device__ void SegSegForceIsotropicCorrGPU(double3 r1, double3 r2, double3 r3, double3 r4, double3 b1, double3 b2,
                                            double3 &fn1, double3 &fn2, double3 &fn3, double3 &fn4)
{
        real8 x1[3], x2[3], x3[3], x4[3], b[3], bp[3];
        real8 f3[3], f4[3];
        real8 vec1[3], vec2[3], t[3], tp[3], tctp[3];
        real8 R[2][3], tempa[2], tempb[2], y[2], z[2];
        int i, j , alt1[3]={1,2,0}, alt2[3]={2,0,1};
        real8 d, c, c2, onemc2, onemc2inv, oneoverL, oneoverLp;
        real8 a2, m4p, m4pd, m8p, m8pd, m4pn, m4pnd, m4pnd2, m4pnd3;
        real8 a2m4pnd, a2m8pd, a2m4pn, a2m8p, a2_d2, a2_d2inv, denom;
        real8 temp1, temp2, temp3, temp4[8], tmp[10];
        real8 yv[4], zv[4], y2[4], z2[4], Ra[4], Rainv[4];
        real8 Ra_Rdot_tp[8], Ra_Rdot_t[8], log_Ra_Rdot_tp[4], log_Ra_Rdot_t[4];
        real8 Ra2_R_tpinv[4], Ra2_R_tinv[4], ylog_Ra_Rdot_tp[4], zlog_Ra_Rdot_t[4];
        real8 yRa2_R_tpinv[4], zRa2_R_tinv[4], y2Ra2_R_tpinv[4], z2Ra2_R_tinv[4];
        real8 adf_003[4], commonf223[4], commonf225[4], commonf025[4], commonf205[4];
        real8 commonf305[4], commonf035[4], ycommonf025[4], zcommonf205[4], zcommonf305[4];
        real8 tf_113[4];
        real8 f_003v[4], f_103v[4], f_013v[4], f_113v[4];
        real8 f_203v[4], f_023v[4], f_005v[4], f_105v[4];
        real8 f_003,  f_103,  f_013,  f_113,  f_203,  f_023,  f_005,  f_105;
        real8 f_015v[4], f_115v[4], f_205v[4], f_025v[4];
        real8 f_215v[4], f_125v[4], f_225v[4], f_305v[4];
        real8 f_015,  f_115,  f_205,  f_025,  f_215,  f_125,  f_225,  f_305;
        real8 f_035v[4], f_315v[4], f_135v[4];
        real8 f_035,  f_315,  f_135;
        real8 Fint_003, Fint_005, Fint_013, Fint_015, Fint_025, Fint_103;
        real8 Fint_105, Fint_115, Fint_125, Fint_205, Fint_215;
        real8 I_003[3], I_005[3], I_013[3], I_015[3], I_025[3], I_103[3];
        real8 I_105[3], I_115[3], I_125[3], I_205[3], I_215[3];
        real8 I00a[3], I01a[3], I10a[3], I00b[3], I01b[3], I10b[3];
        real8 bctctp[3], bct[3], bpctpct[3], bpctp[3], tcbpct[3];
        real8 bctdbp, bpctpdb, tcbpdb, tcbpdtp, tpcbdbp;
        real8 tctpct[3], tpct[3];
        real8 tctpcbpdb, tctpcbpdtp, tctpdb, tdb, tdbp;
        real8 tpcbctp[3], tpctctp[3];
        real8 tpcbdt, tpctcbdbp, tpctcbdt, tpctdbp, tpdb, tpdbp;
        real8 pivalue=3.141592653589793;           
        
        fn1.x = 0.0;
        fn1.y = 0.0;
        fn1.z = 0.0;

        fn2.x = 0.0;
        fn2.y = 0.0;
        fn2.z = 0.0;

        fn3.x = 0.0;
        fn3.y = 0.0;
        fn3.z = 0.0;

        fn4.x = 0.0;
        fn4.y = 0.0;
        fn4.z = 0.0;

        x1[0]=r1.x;
        x1[1]=r1.y;
        x1[2]=r1.z;
        x2[0]=r2.x;
        x2[1]=r2.y;
        x2[2]=r2.z;
        x3[0]=r3.x;
        x3[1]=r3.y;
        x3[2]=r3.z;
        x4[0]=r4.x;
        x4[1]=r4.y;
        x4[2]=r4.z;
        
        b[0]=b2.x;
        b[1]=b2.y;
        b[2]=b2.z;
        bp[0]=b1.x;
        bp[1]=b1.y;
        bp[2]=b1.z;
        
        #pragma unroll   
        for(i=0;i<3;i++) { 
            vec1[i]=x4[i]-x3[i];
            vec2[i]=x2[i]-x1[i];
        }

        temp1=0.0e0;
        temp2=0.0e0;    
        
        #pragma unroll 
        for(i=0;i<3;i++) { 
            temp1+=vec1[i]*vec1[i];
            temp2+=vec2[i]*vec2[i];
        }

        oneoverL =1/sqrt(temp1);
        oneoverLp=1/sqrt(temp2);
        
        #pragma unroll         
        for(i=0;i<3;i++) { 
            t[i]=vec1[i]*oneoverL;
            tp[i]=vec2[i]*oneoverLp;
        }
        
        c=0.0e0;
        
        #pragma unroll 
        for(i=0;i<3;i++) { 
            c+=t[i]*tp[i];
        }
                 
        c2=c*c;
        onemc2=1-c2;

        {	
			#pragma unroll 
            for(i=0;i<3;i++) {
                tctp[i]=t[alt1[i]]*tp[alt2[i]]-t[alt2[i]]*tp[alt1[i]];
            }

            onemc2inv = 1/onemc2;
            
            #pragma unroll 
            for(i=0;i<3;i++) { 
                R[0][i]=x3[i]-x1[i];
                R[1][i]=x4[i]-x2[i];
            }

            d=0.0e0;
            
            #pragma unroll
            for (j=0;j<2;j++) { 
                tempa[j]=0.0e0;
                tempb[j]=0.0e0;
            }
            
            #pragma unroll
            for(i=0;i<3;i++) { 
                d+=0.5e0*((x4[i]+x3[i])-(x2[i]+x1[i]))*tctp[i];
                for (j=0;j<2;j++) { 
                    tempa[j]+=R[j][i]*t[i];
                    tempb[j]+=R[j][i]*tp[i];
                }
            }

            d*=onemc2inv;
            
            #pragma unroll
            for (j=0;j<2;j++) { 
                y[j]=(tempa[j]-c*tempb[j])*onemc2inv;
                z[j]=(tempb[j]-c*tempa[j])*onemc2inv;
            }

/*          now we calculate the definite integrals of the force calculation  */

            #pragma unroll
            for (j=0;j<2;j++) {
                yv[2*j]=y[j];
                yv[2*j+1]=y[j];
                zv[j]=z[j];
                zv[j+2]=z[j];
            }
            
            a2_d2 = a*a+d*d*onemc2;
            
            #pragma unroll
            for (j=0;j<4;j++) {
                y2[j] = yv[j]*yv[j];
                z2[j] = zv[j]*zv[j];
                
            }
            
            #pragma unroll
            for (j=0;j<4;j++) {
                temp4[j]=a2_d2 + y2[j] + z2[j] + 2.0e0*yv[j]*zv[j]*c;
            }

            temp1=onemc2*a2_d2;
            
            #pragma unroll
            for (j=0;j<4;j++) {
                Ra[j]=sqrt(temp4[j]);
            }

            temp2=sqrt(temp1);
            
            #pragma unroll
            for (j=0;j<4;j++) {
                Rainv[j]=1.0e0/Ra[j];
            }

            denom=1.0e0/temp2;
            a2_d2inv=1.0e0/a2_d2;
            
            #pragma unroll
            for (j=0;j<4;j++) {
                Ra_Rdot_tp[j] = Ra[j]+(zv[j]+yv[j]*c);       
                Ra_Rdot_t[j]  = Ra[j]+(yv[j]+zv[j]*c);
				Ra_Rdot_tp[j+4] = Ra[j]-(zv[j]+yv[j]*c);       
                Ra_Rdot_t[j+4]  = Ra[j]-(yv[j]+zv[j]*c); 
            }
            
            #pragma unroll
            for (j=0;j<4;j++) {
                log_Ra_Rdot_tp[j] =0.5e0*(log(Ra_Rdot_tp[j])-log(Ra_Rdot_tp[j+4]));
                log_Ra_Rdot_t[j]  =0.5e0*(log(Ra_Rdot_t[j])-log(Ra_Rdot_t[j+4]));
            }
            
            #pragma unroll
            for (j=0;j<4;j++) {
                Ra2_R_tpinv[j] = 0.5e0*(Rainv[j]/Ra_Rdot_tp[j]- Rainv[j]/Ra_Rdot_tp[j+4]);
                Ra2_R_tinv[j] =  0.5e0*(Rainv[j]/Ra_Rdot_t[j]- Rainv[j]/Ra_Rdot_t[j+4]);
            }
            
            #pragma unroll
            for (j=0;j<4;j++) {
                ylog_Ra_Rdot_tp[j] = yv[j]*log_Ra_Rdot_tp[j];
                yRa2_R_tpinv[j]    = yv[j]*   Ra2_R_tpinv[j];
                zlog_Ra_Rdot_t[j]  = zv[j]*log_Ra_Rdot_t[j];
                zRa2_R_tinv[j]     = zv[j]*   Ra2_R_tinv[j];
                
            }
            
            #pragma unroll
            for (j=0;j<4;j++) {
                y2Ra2_R_tpinv[j] = yv[j]* yRa2_R_tpinv[j];
                z2Ra2_R_tinv[j]  = zv[j]*  zRa2_R_tinv[j];
            }

            temp1=denom*(1+c);
            
            #pragma unroll
            for (j=0;j<4;j++) {
                temp4[j]=temp1*(Ra[j]+(yv[j]+zv[j]));
				temp4[j+4]=temp1*(Ra[j]-(yv[j]+zv[j]));
            }
            
            #pragma unroll
            for (j=0;j<4;j++) {
                f_003v[j]=0.5e0*(atan(temp4[j])+atan(temp4[j+4]));
            }
            
            temp1=-2.0e0*denom;
            
            #pragma unroll
            for (j=0;j<4;j++) {
                f_003v[j]*=temp1;
            }
            
            #pragma unroll
            for (j=0;j<4;j++) {
                adf_003[j]=f_003v[j]*a2_d2;
            }
            
            #pragma unroll
            for (j=0;j<4;j++) {
                commonf223[j] = c*Ra[j] - adf_003[j];
                f_103v[j] = c*log_Ra_Rdot_t[j]  - log_Ra_Rdot_tp[j];
                f_013v[j] = c*log_Ra_Rdot_tp[j] - log_Ra_Rdot_t [j];
                f_113v[j] = c*adf_003[j] - Ra[j];
            }
            
            #pragma unroll
            for (j=0;j<4;j++) {
                commonf223[j] *= onemc2inv;
                f_103v[j] *=      onemc2inv;
                f_013v[j] *=      onemc2inv;
                f_113v[j] *=      onemc2inv;
            }
            
            #pragma unroll
            for (j=0;j<4;j++) {
                commonf225[j] = f_003v[j] - c*Rainv[j];
                commonf025[j] = c*yRa2_R_tpinv[j] - Rainv[j];
                commonf205[j] = c*zRa2_R_tinv[j]  - Rainv[j];
                commonf305[j] = log_Ra_Rdot_t[j]  -(yv[j]-c*zv[j])*Rainv[j] - c2*z2Ra2_R_tinv[j];
                commonf035[j] = log_Ra_Rdot_tp[j] -(zv[j]-c*yv[j])*Rainv[j] - c2*y2Ra2_R_tpinv[j]; 
                f_203v[j] =  zlog_Ra_Rdot_t[j]  + commonf223[j];
                f_023v[j] =  ylog_Ra_Rdot_tp[j] + commonf223[j];
                f_005v[j] = f_003v[j] - yRa2_R_tpinv[j] - zRa2_R_tinv[j];
                f_105v[j] = Ra2_R_tpinv[j] - c*Ra2_R_tinv[j];
                f_015v[j] = Ra2_R_tinv[j]  - c*Ra2_R_tpinv[j];
                f_115v[j] = Rainv[j] - c*(yRa2_R_tpinv[j] + zRa2_R_tinv[j] + f_003v[j]);
            }
            
            #pragma unroll
            for (j=0;j<4;j++) {
                ycommonf025[j] = yv[j]*commonf025[j];
                zcommonf205[j] = zv[j]*commonf205[j];
                zcommonf305[j] = zv[j]*commonf305[j];
                tf_113[j]=2.0e0*f_113v[j];
                f_205v[j] = yRa2_R_tpinv[j] + c2*zRa2_R_tinv[j]  + commonf225[j];
                f_025v[j] = zRa2_R_tinv[j]  + c2*yRa2_R_tpinv[j] + commonf225[j];
                f_305v[j] = y2Ra2_R_tpinv[j] + c*commonf305[j] + 2.0e0*f_103v[j];
                f_035v[j] = z2Ra2_R_tinv[j]  + c*commonf035[j] + 2.0e0*f_013v[j];
            }
            
            #pragma unroll
            for (j=0;j<4;j++) {
                f_215v[j] = f_013v[j] - ycommonf025[j] + c*(zcommonf205[j]-f_103v[j]); 
                f_125v[j] = f_103v[j] - zcommonf205[j] + c*(ycommonf025[j] - f_013v[j]); 
                f_225v[j] = f_203v[j] - zcommonf305[j] + c*(y2[j]*commonf025[j] - tf_113[j]);
                f_315v[j] = tf_113[j] - y2[j]*commonf025[j] + c*(zcommonf305[j] - f_203v[j]);
                f_135v[j] = tf_113[j] - z2[j]*commonf205[j] + c*(yv[j]*commonf035[j]-f_023v[j]);
            }
            
             
            f_003= (f_003v[0]+f_003v[3])-(f_003v[1]+f_003v[2]);
            f_013= (f_013v[0]+f_013v[3])-(f_013v[1]+f_013v[2]);
            f_103= (f_103v[0]+f_103v[3])-(f_103v[1]+f_103v[2]);
            f_113= (f_113v[0]+f_113v[3])-(f_113v[1]+f_113v[2]);
            f_023= (f_023v[0]+f_023v[3])-(f_023v[1]+f_023v[2]);
            f_203= (f_203v[0]+f_203v[3])-(f_203v[1]+f_203v[2]);
            f_005= (f_005v[0]+f_005v[3])-(f_005v[1]+f_005v[2]);
            f_015= (f_015v[0]+f_015v[3])-(f_015v[1]+f_015v[2]);
            f_105= (f_105v[0]+f_105v[3])-(f_105v[1]+f_105v[2]);
            f_115= (f_115v[0]+f_115v[3])-(f_115v[1]+f_115v[2]);
            f_025= (f_025v[0]+f_025v[3])-(f_025v[1]+f_025v[2]);
            f_205= (f_205v[0]+f_205v[3])-(f_205v[1]+f_205v[2]);
            f_215= (f_215v[0]+f_215v[3])-(f_215v[1]+f_215v[2]);
            f_125= (f_125v[0]+f_125v[3])-(f_125v[1]+f_125v[2]);
            f_035= (f_035v[0]+f_035v[3])-(f_035v[1]+f_035v[2]);
            f_305= (f_305v[0]+f_305v[3])-(f_305v[1]+f_305v[2]);
            f_225= (f_225v[0]+f_225v[3])-(f_225v[1]+f_225v[2]);
            f_135= (f_135v[0]+f_135v[3])-(f_135v[1]+f_135v[2]);
            f_315= (f_315v[0]+f_315v[3])-(f_315v[1]+f_315v[2]);
            
            
            f_005 *= a2_d2inv;
            f_105 *= onemc2inv;
            f_015 *= onemc2inv;
            f_115 *= onemc2inv;
            f_205 *= onemc2inv;
            f_025 *= onemc2inv;
            f_305 *= onemc2inv;
            f_035 *= onemc2inv;            
            f_215 *= onemc2inv; 
            f_125 *= onemc2inv; 
            f_225 *= onemc2inv;
            f_315 *= onemc2inv;
            f_135 *= onemc2inv;
            
      
/* now construct the vector coefficients for the definite integrals */

            a2 = a*a;
            m4p = 0.25 * MU / pivalue;
            m4pd =  m4p * d;
            m8p = 0.5 * m4p;
            m8pd = m8p * d;
            m4pn = m4p / ( 1 - NU );
            m4pnd = m4pn * d;
            m4pnd2 = m4pnd * d;
            m4pnd3 = m4pnd2 * d;
            a2m4pnd = a2 * m4pnd;
            a2m8pd = a2 * m8pd;
            a2m4pn = a2 * m4pn;
            a2m8p = a2 * m8p;

            #pragma unroll
            for (i=0;i<3;i++) {
                tpct[i]=-tctp[i];
                bct[i]=b[alt1[i]]*t[alt2[i]]-b[alt2[i]]*t[alt1[i]];
                bpctp[i]=bp[alt1[i]]*tp[alt2[i]]-bp[alt2[i]]*tp[alt1[i]];
                
            }

            tdb=0.0e0;
            tdbp=0.0e0;
            tpdb=0.0e0;
            tpdbp=0.0e0;
            tctpdb=0.0e0;
            tpctdbp=0.0e0;
            bpctpdb=0.0e0;
            bctdbp=0.0e0;
            
            #pragma unroll
            for (i=0;i<3;i++) {
                tdb    +=t[i]*b[i];
                tdbp   +=t[i]*bp[i];
                tpdb   +=tp[i]*b[i];
                tpdbp  +=tp[i]*bp[i];
                tctpdb +=tctp[i]*b[i];
                tpctdbp+=tpct[i]*bp[i];
                bpctpdb+=bpctp[i]*b[i];
                bctdbp +=bct[i]*bp[i];
            }
            
            #pragma unroll
            for (i=0;i<3;i++) {
                tctpct[i]    =        tp[i] -     c*t[i];
                tpctctp[i]   =         t[i] -    c*tp[i];
                tcbpct[i]    =        bp[i] -  tdbp*t[i];
                tpcbctp[i]   =         b[i] - tpdb*tp[i];
                bpctpct[i]   =   tdbp*tp[i] -    c*bp[i];
                bctctp[i]    =    tpdb*t[i] -     c*b[i];
            }
                
            
            tctpcbpdtp = tdbp - tpdbp*c;
            tpctcbdt = tpdb - tdb*c;
            tctpcbpdb =  tdbp*tpdb - tpdbp*tdb;
            tpctcbdbp = tctpcbpdb;
            tcbpdtp = tpctdbp; 
            tpcbdt = tctpdb;
            tcbpdb = bctdbp;
            tpcbdbp = bpctpdb;

/*
 *          Only calculate the forces for segment p3->p4 if at least one
 *          of the segment's nodes is local to the current domain.
 */
            //if (seg34Local) {

                temp1 = tdbp*tpdb + tctpcbpdb;
                
                #pragma unroll
                for (i=0;i<3;i++) {
                    I00a[i] = temp1 * tpct[i];
                    I00b[i] = tctpcbpdtp * bct[i];
                }

                temp1 = (m4pnd * tctpdb);
                temp2 = (m4pnd * bpctpdb);
                temp3 = (m4pnd3 * tctpcbpdtp*tctpdb);
                
                #pragma unroll
                for (i=0;i<3;i++) {
                    I_003[i] = m4pd*I00a[i] - m4pnd*I00b[i] + temp1*bpctpct[i] +
                            temp2*tctpct[i]; 
                    I_005[i] = a2m8pd*I00a[i] - a2m4pnd*I00b[i] - temp3*tctpct[i];
                    I10a[i] = tcbpct[i]*tpdb - tctp[i]*tcbpdb;
                    I10b[i] = bct[i] * tcbpdtp;
                    
                }

                temp1 = (m4pn * tdb);
                temp2 = m4pnd2 * (tcbpdtp*tctpdb + tctpcbpdtp*tdb);
                
                #pragma unroll
                for (i=0;i<3;i++) {
                    I_103[i] = temp1*bpctpct[i] + m4p*I10a[i] - m4pn*I10b[i];
                    I_105[i] = a2m8p*I10a[i] - a2m4pn*I10b[i] - temp2*tctpct[i];
                    I01a[i] = tctp[i]*bpctpdb - bpctpct[i]*tpdb;
                }

                tmp[0] = (m4pn * tpdb); 
                tmp[1] = (m4pn * bpctpdb);
                tmp[2] = (m4pnd2 * tctpcbpdtp * tpdb);
                tmp[3] = (m4pnd2 * tctpcbpdtp * tctpdb);
                tmp[4] = (m4pnd * tcbpdtp * tdb);
                tmp[5] = (m4pnd * tctpcbpdtp * tpdb) ;
                tmp[6] = (m4pnd * (tctpcbpdtp*tdb + tcbpdtp*tctpdb));
                tmp[7] = (m4pnd * tcbpdtp * tpdb);
                tmp[8] = (m4pn * tcbpdtp * tdb);
                tmp[9] = (m4pn * tcbpdtp * tpdb);
                
                #pragma unroll
                for (i=0;i<3;i++) {
                    I_013[i] = m4p*I01a[i] + tmp[0]*bpctpct[i] - tmp[1]*tctp[i];
                    I_015[i] = a2m8p*I01a[i] - tmp[2]*tctpct[i] + tmp[3]*tctp[i];
                    I_205[i] = -tmp[4] * tctpct[i];
                    I_025[i] = tmp[5] * tctp[i]; 
                    I_115[i] =  tmp[6]*tctp[i] - tmp[7]*tctpct[i];
                    I_215[i] = tmp[8] * tctp[i];
                    I_125[i] = tmp[9] * tctp[i];
                }
  
                Fint_003 = f_103 - y[0]*f_003;
                Fint_103 = f_203 - y[0]*f_103;
                Fint_013 = f_113 - y[0]*f_013;
                Fint_005 = f_105 - y[0]*f_005;
                Fint_105 = f_205 - y[0]*f_105;
                Fint_015 = f_115 - y[0]*f_015;
                Fint_115 = f_215 - y[0]*f_115;
                Fint_205 = f_305 - y[0]*f_205;
                Fint_025 = f_125 - y[0]*f_025;
                Fint_215 = f_315 - y[0]*f_215;
                Fint_125 = f_225 - y[0]*f_125;
                
                #pragma unroll
                for (i=0;i<3;i++) {
                    f4[i]=(I_003[i]*Fint_003 + I_103[i]*Fint_103 + I_013[i]*Fint_013 +
                           I_005[i]*Fint_005 + I_105[i]*Fint_105 + I_015[i]*Fint_015 +
                           I_115[i]*Fint_115 + I_205[i]*Fint_205 + I_025[i]*Fint_025 +
                           I_215[i]*Fint_215 + I_125[i]*Fint_125) * oneoverL;
                }

                Fint_003 = y[1]*f_003 - f_103;
                Fint_103 = y[1]*f_103 - f_203;
                Fint_013 = y[1]*f_013 - f_113;
                Fint_005 = y[1]*f_005 - f_105;
                Fint_105 = y[1]*f_105 - f_205;
                Fint_015 = y[1]*f_015 - f_115;
                Fint_115 = y[1]*f_115 - f_215;
                Fint_205 = y[1]*f_205 - f_305;
                Fint_025 = y[1]*f_025 - f_125;
                Fint_215 = y[1]*f_215 - f_315;
                Fint_125 = y[1]*f_125 - f_225;
                
                #pragma unroll
                for (i=0;i<3;i++) {
                    f3[i]=(I_003[i]*Fint_003 + I_103[i]*Fint_103 + I_013[i]*Fint_013 +
                           I_005[i]*Fint_005 + I_105[i]*Fint_105 + I_015[i]*Fint_015 +
                           I_115[i]*Fint_115 + I_205[i]*Fint_205 + I_025[i]*Fint_025 +
                           I_215[i]*Fint_215 + I_125[i]*Fint_125) * oneoverL;
                }

                fn3.x=f3[0];
                fn3.y=f3[1];
                fn3.z=f3[2];
                fn4.x=f4[0];
                fn4.y=f4[1];
                fn4.z=f4[2];

            //} /* if segment p3->p4 is "local" */
       }
       
       return;
}

/*------------------------------------------------------------------------
 *
 *      Function:    SegSegForceIsotropicGPU
 *
 *-----------------------------------------------------------------------*/
__device__ void SegSegForceIsotropicGPU(double3 r1, double3 r2, double3 r3, double3 r4, double3 b1, double3 b2,
                                        double3 &fn1, double3 &fn2, double3 &fn3, double3 &fn4)
{
        real8 x1[3], x2[3], x3[3], x4[3], b[3], bp[3];
        real8 f1[3], f2[3], f3[3], f4[3];
        real8 vec1[3], vec2[3], t[3], tp[3], tctp[3];
        real8 R[2][3], tempa[2], tempb[2], y[2], z[2];
        int i, j , alt1[3]={1,2,0}, alt2[3]={2,0,1};
        real8 eps, d, c, c2, onemc2, onemc2inv, oneoverL, oneoverLp;
        real8 a2, m4p, m4pd, m8p, m8pd, m4pn, m4pnd, m4pnd2, m4pnd3;
        real8 a2m4pnd, a2m8pd, a2m4pn, a2m8p, a2_d2, a2_d2inv, denom;
        real8 temp1, temp2, temp3, temp4[8], tmp[10];
        real8 yv[4], zv[4], y2[4], z2[4], Ra[4], Rainv[4];
        real8 Ra_Rdot_tp[8], Ra_Rdot_t[8], log_Ra_Rdot_tp[4], log_Ra_Rdot_t[4];
        real8 Ra2_R_tpinv[4], Ra2_R_tinv[4], ylog_Ra_Rdot_tp[4], zlog_Ra_Rdot_t[4];
        real8 yRa2_R_tpinv[4], zRa2_R_tinv[4], y2Ra2_R_tpinv[4], z2Ra2_R_tinv[4];
        real8 adf_003[4], commonf223[4], commonf225[4], commonf025[4], commonf205[4];
        real8 commonf305[4], commonf035[4], ycommonf025[4], zcommonf205[4], zcommonf305[4];
        real8 tf_113[4];
        real8 f_003v[4], f_103v[4], f_013v[4], f_113v[4];
        real8 f_203v[4], f_023v[4], f_005v[4], f_105v[4];
        real8 f_003,  f_103,  f_013,  f_113,  f_203,  f_023,  f_005,  f_105;
        real8 f_015v[4], f_115v[4], f_205v[4], f_025v[4];
        real8 f_215v[4], f_125v[4], f_225v[4], f_305v[4];
        real8 f_015,  f_115,  f_205,  f_025,  f_215,  f_125,  f_225,  f_305;
        real8 f_035v[4], f_315v[4], f_135v[4];
        real8 f_035,  f_315,  f_135;
        real8 Fint_003, Fint_005, Fint_013, Fint_015, Fint_025, Fint_103;
        real8 Fint_105, Fint_115, Fint_125, Fint_205, Fint_215;
        real8 I_003[3], I_005[3], I_013[3], I_015[3], I_025[3], I_103[3];
        real8 I_105[3], I_115[3], I_125[3], I_205[3], I_215[3];
        real8 I00a[3], I01a[3], I10a[3], I00b[3], I01b[3], I10b[3];
        real8 bctctp[3], bct[3], bpctpct[3], bpctp[3], tcbpct[3];
        real8 bctdbp, bpctpdb, tcbpdb, tcbpdtp, tpcbdbp;
        real8 tctpct[3], tpct[3];
        real8 tctpcbpdb, tctpcbpdtp, tctpdb, tdb, tdbp;
        real8 tpcbctp[3], tpctctp[3];
        real8 tpcbdt, tpctcbdbp, tpctcbdt, tpctdbp, tpdb, tpdbp;
        real8 pivalue=3.141592653589793;

        eps = 1e-4;            
        
        fn1.x = 0.0;
        fn1.y = 0.0;
        fn1.z = 0.0;

        fn2.x = 0.0;
        fn2.y = 0.0;
        fn2.z = 0.0;

        fn3.x = 0.0;
        fn3.y = 0.0;
        fn3.z = 0.0;

        fn4.x = 0.0;
        fn4.y = 0.0;
        fn4.z = 0.0;

        x1[0]=r1.x;
        x1[1]=r1.y;
        x1[2]=r1.z;
        x2[0]=r2.x;
        x2[1]=r2.y;
        x2[2]=r2.z;
        x3[0]=r3.x;
        x3[1]=r3.y;
        x3[2]=r3.z;
        x4[0]=r4.x;
        x4[1]=r4.y;
        x4[2]=r4.z;
        
        b[0]=b2.x;
        b[1]=b2.y;
        b[2]=b2.z;
        bp[0]=b1.x;
        bp[1]=b1.y;
        bp[2]=b1.z;
        
        #pragma unroll   
        for(i=0;i<3;i++) { 
            vec1[i]=x4[i]-x3[i];
            vec2[i]=x2[i]-x1[i];
        }

        temp1=0.0e0;
        temp2=0.0e0;    
        
        #pragma unroll 
        for(i=0;i<3;i++) { 
            temp1+=vec1[i]*vec1[i];
            temp2+=vec2[i]*vec2[i];
        }

        oneoverL =1/sqrt(temp1);
        oneoverLp=1/sqrt(temp2);
        
        #pragma unroll         
        for(i=0;i<3;i++) { 
            t[i]=vec1[i]*oneoverL;
            tp[i]=vec2[i]*oneoverLp;
        }
        
        c=0.0e0;
        
        #pragma unroll 
        for(i=0;i<3;i++) { 
            c+=t[i]*tp[i];
        }
                 
        c2=c*c;
        onemc2=1-c2;
        
        if (onemc2 > eps) {
			
			#pragma unroll 
            for(i=0;i<3;i++) {
                tctp[i]=t[alt1[i]]*tp[alt2[i]]-t[alt2[i]]*tp[alt1[i]];
            }

            onemc2inv = 1/onemc2;
            
            #pragma unroll 
            for(i=0;i<3;i++) { 
                R[0][i]=x3[i]-x1[i];
                R[1][i]=x4[i]-x2[i];
            }

            d=0.0e0;
            
            #pragma unroll
            for (j=0;j<2;j++) { 
                tempa[j]=0.0e0;
                tempb[j]=0.0e0;
            }
            
            #pragma unroll
            for(i=0;i<3;i++) { 
                d+=0.5e0*((x4[i]+x3[i])-(x2[i]+x1[i]))*tctp[i];
                for (j=0;j<2;j++) { 
                    tempa[j]+=R[j][i]*t[i];
                    tempb[j]+=R[j][i]*tp[i];
                }
            }

            d*=onemc2inv;
            
            #pragma unroll
            for (j=0;j<2;j++) { 
                y[j]=(tempa[j]-c*tempb[j])*onemc2inv;
                z[j]=(tempb[j]-c*tempa[j])*onemc2inv;
            }

/*          now we calculate the definite integrals of the force calculation  */

            #pragma unroll
            for (j=0;j<2;j++) {
                yv[2*j]=y[j];
                yv[2*j+1]=y[j];
                zv[j]=z[j];
                zv[j+2]=z[j];
            }
            
            a2_d2 = a*a+d*d*onemc2;
            
            #pragma unroll
            for (j=0;j<4;j++) {
                y2[j] = yv[j]*yv[j];
                z2[j] = zv[j]*zv[j];
                
            }
            
            #pragma unroll
            for (j=0;j<4;j++) {
                temp4[j]=a2_d2 + y2[j] + z2[j] + 2.0e0*yv[j]*zv[j]*c;
            }

            temp1=onemc2*a2_d2;
            
            #pragma unroll
            for (j=0;j<4;j++) {
                Ra[j]=sqrt(temp4[j]);
            }

            temp2=sqrt(temp1);
            
            #pragma unroll
            for (j=0;j<4;j++) {
                Rainv[j]=1.0e0/Ra[j];
            }

            denom=1.0e0/temp2;
            a2_d2inv=1.0e0/a2_d2;
            
            #pragma unroll
            for (j=0;j<4;j++) {
                Ra_Rdot_tp[j] = Ra[j]+(zv[j]+yv[j]*c);       
                Ra_Rdot_t[j]  = Ra[j]+(yv[j]+zv[j]*c);
				Ra_Rdot_tp[j+4] = Ra[j]-(zv[j]+yv[j]*c);       
                Ra_Rdot_t[j+4]  = Ra[j]-(yv[j]+zv[j]*c); 
            }
            
            #pragma unroll
            for (j=0;j<4;j++) {
                log_Ra_Rdot_tp[j] =0.5e0*(log(Ra_Rdot_tp[j])-log(Ra_Rdot_tp[j+4]));
                log_Ra_Rdot_t[j]  =0.5e0*(log(Ra_Rdot_t[j])-log(Ra_Rdot_t[j+4]));
            }
            
            #pragma unroll
            for (j=0;j<4;j++) {
                Ra2_R_tpinv[j] = 0.5e0*(Rainv[j]/Ra_Rdot_tp[j]- Rainv[j]/Ra_Rdot_tp[j+4]);
                Ra2_R_tinv[j] =  0.5e0*(Rainv[j]/Ra_Rdot_t[j]- Rainv[j]/Ra_Rdot_t[j+4]);
            }
            
            #pragma unroll
            for (j=0;j<4;j++) {
                ylog_Ra_Rdot_tp[j] = yv[j]*log_Ra_Rdot_tp[j];
                yRa2_R_tpinv[j]    = yv[j]*   Ra2_R_tpinv[j];
                zlog_Ra_Rdot_t[j]  = zv[j]*log_Ra_Rdot_t[j];
                zRa2_R_tinv[j]     = zv[j]*   Ra2_R_tinv[j];
                
            }
            
            #pragma unroll
            for (j=0;j<4;j++) {
                y2Ra2_R_tpinv[j] = yv[j]* yRa2_R_tpinv[j];
                z2Ra2_R_tinv[j]  = zv[j]*  zRa2_R_tinv[j];
            }

            temp1=denom*(1+c);
            
            #pragma unroll
            for (j=0;j<4;j++) {
                temp4[j]=temp1*(Ra[j]+(yv[j]+zv[j]));
				temp4[j+4]=temp1*(Ra[j]-(yv[j]+zv[j]));
            }
            
            #pragma unroll
            for (j=0;j<4;j++) {
                f_003v[j]=0.5e0*(atan(temp4[j])+atan(temp4[j+4]));
            }
            
            temp1=-2.0e0*denom;
            
            #pragma unroll
            for (j=0;j<4;j++) {
                f_003v[j]*=temp1;
            }
            
            #pragma unroll
            for (j=0;j<4;j++) {
                adf_003[j]=f_003v[j]*a2_d2;
            }
            
            #pragma unroll
            for (j=0;j<4;j++) {
                commonf223[j] = c*Ra[j] - adf_003[j];
                f_103v[j] = c*log_Ra_Rdot_t[j]  - log_Ra_Rdot_tp[j];
                f_013v[j] = c*log_Ra_Rdot_tp[j] - log_Ra_Rdot_t [j];
                f_113v[j] = c*adf_003[j] - Ra[j];
            }
            
            #pragma unroll
            for (j=0;j<4;j++) {
                commonf223[j] *= onemc2inv;
                f_103v[j] *=      onemc2inv;
                f_013v[j] *=      onemc2inv;
                f_113v[j] *=      onemc2inv;
            }
            
            #pragma unroll
            for (j=0;j<4;j++) {
                commonf225[j] = f_003v[j] - c*Rainv[j];
                commonf025[j] = c*yRa2_R_tpinv[j] - Rainv[j];
                commonf205[j] = c*zRa2_R_tinv[j]  - Rainv[j];
                commonf305[j] = log_Ra_Rdot_t[j]  -(yv[j]-c*zv[j])*Rainv[j] - c2*z2Ra2_R_tinv[j];
                commonf035[j] = log_Ra_Rdot_tp[j] -(zv[j]-c*yv[j])*Rainv[j] - c2*y2Ra2_R_tpinv[j]; 
                f_203v[j] =  zlog_Ra_Rdot_t[j]  + commonf223[j];
                f_023v[j] =  ylog_Ra_Rdot_tp[j] + commonf223[j];
                f_005v[j] = f_003v[j] - yRa2_R_tpinv[j] - zRa2_R_tinv[j];
                f_105v[j] = Ra2_R_tpinv[j] - c*Ra2_R_tinv[j];
                f_015v[j] = Ra2_R_tinv[j]  - c*Ra2_R_tpinv[j];
                f_115v[j] = Rainv[j] - c*(yRa2_R_tpinv[j] + zRa2_R_tinv[j] + f_003v[j]);
            }
            
            #pragma unroll
            for (j=0;j<4;j++) {
                ycommonf025[j] = yv[j]*commonf025[j];
                zcommonf205[j] = zv[j]*commonf205[j];
                zcommonf305[j] = zv[j]*commonf305[j];
                tf_113[j]=2.0e0*f_113v[j];
                f_205v[j] = yRa2_R_tpinv[j] + c2*zRa2_R_tinv[j]  + commonf225[j];
                f_025v[j] = zRa2_R_tinv[j]  + c2*yRa2_R_tpinv[j] + commonf225[j];
                f_305v[j] = y2Ra2_R_tpinv[j] + c*commonf305[j] + 2.0e0*f_103v[j];
                f_035v[j] = z2Ra2_R_tinv[j]  + c*commonf035[j] + 2.0e0*f_013v[j];
            }
            
            #pragma unroll
            for (j=0;j<4;j++) {
                f_215v[j] = f_013v[j] - ycommonf025[j] + c*(zcommonf205[j]-f_103v[j]); 
                f_125v[j] = f_103v[j] - zcommonf205[j] + c*(ycommonf025[j] - f_013v[j]); 
                f_225v[j] = f_203v[j] - zcommonf305[j] + c*(y2[j]*commonf025[j] - tf_113[j]);
                f_315v[j] = tf_113[j] - y2[j]*commonf025[j] + c*(zcommonf305[j] - f_203v[j]);
                f_135v[j] = tf_113[j] - z2[j]*commonf205[j] + c*(yv[j]*commonf035[j]-f_023v[j]);
            }
            
             
            f_003= (f_003v[0]+f_003v[3])-(f_003v[1]+f_003v[2]);
            f_013= (f_013v[0]+f_013v[3])-(f_013v[1]+f_013v[2]);
            f_103= (f_103v[0]+f_103v[3])-(f_103v[1]+f_103v[2]);
            f_113= (f_113v[0]+f_113v[3])-(f_113v[1]+f_113v[2]);
            f_023= (f_023v[0]+f_023v[3])-(f_023v[1]+f_023v[2]);
            f_203= (f_203v[0]+f_203v[3])-(f_203v[1]+f_203v[2]);
            f_005= (f_005v[0]+f_005v[3])-(f_005v[1]+f_005v[2]);
            f_015= (f_015v[0]+f_015v[3])-(f_015v[1]+f_015v[2]);
            f_105= (f_105v[0]+f_105v[3])-(f_105v[1]+f_105v[2]);
            f_115= (f_115v[0]+f_115v[3])-(f_115v[1]+f_115v[2]);
            f_025= (f_025v[0]+f_025v[3])-(f_025v[1]+f_025v[2]);
            f_205= (f_205v[0]+f_205v[3])-(f_205v[1]+f_205v[2]);
            f_215= (f_215v[0]+f_215v[3])-(f_215v[1]+f_215v[2]);
            f_125= (f_125v[0]+f_125v[3])-(f_125v[1]+f_125v[2]);
            f_035= (f_035v[0]+f_035v[3])-(f_035v[1]+f_035v[2]);
            f_305= (f_305v[0]+f_305v[3])-(f_305v[1]+f_305v[2]);
            f_225= (f_225v[0]+f_225v[3])-(f_225v[1]+f_225v[2]);
            f_135= (f_135v[0]+f_135v[3])-(f_135v[1]+f_135v[2]);
            f_315= (f_315v[0]+f_315v[3])-(f_315v[1]+f_315v[2]);
            
            
            f_005 *= a2_d2inv;
            f_105 *= onemc2inv;
            f_015 *= onemc2inv;
            f_115 *= onemc2inv;
            f_205 *= onemc2inv;
            f_025 *= onemc2inv;
            f_305 *= onemc2inv;
            f_035 *= onemc2inv;            
            f_215 *= onemc2inv; 
            f_125 *= onemc2inv; 
            f_225 *= onemc2inv;
            f_315 *= onemc2inv;
            f_135 *= onemc2inv;
            
      
/* now construct the vector coefficients for the definite integrals */

            a2 = a*a;
            m4p = 0.25 * MU / pivalue;
            m4pd =  m4p * d;
            m8p = 0.5 * m4p;
            m8pd = m8p * d;
            m4pn = m4p / ( 1 - NU );
            m4pnd = m4pn * d;
            m4pnd2 = m4pnd * d;
            m4pnd3 = m4pnd2 * d;
            a2m4pnd = a2 * m4pnd;
            a2m8pd = a2 * m8pd;
            a2m4pn = a2 * m4pn;
            a2m8p = a2 * m8p;

            #pragma unroll
            for (i=0;i<3;i++) {
                tpct[i]=-tctp[i];
                bct[i]=b[alt1[i]]*t[alt2[i]]-b[alt2[i]]*t[alt1[i]];
                bpctp[i]=bp[alt1[i]]*tp[alt2[i]]-bp[alt2[i]]*tp[alt1[i]];
                
            }

            tdb=0.0e0;
            tdbp=0.0e0;
            tpdb=0.0e0;
            tpdbp=0.0e0;
            tctpdb=0.0e0;
            tpctdbp=0.0e0;
            bpctpdb=0.0e0;
            bctdbp=0.0e0;
            
            #pragma unroll
            for (i=0;i<3;i++) {
                tdb    +=t[i]*b[i];
                tdbp   +=t[i]*bp[i];
                tpdb   +=tp[i]*b[i];
                tpdbp  +=tp[i]*bp[i];
                tctpdb +=tctp[i]*b[i];
                tpctdbp+=tpct[i]*bp[i];
                bpctpdb+=bpctp[i]*b[i];
                bctdbp +=bct[i]*bp[i];
            }
            
            #pragma unroll
            for (i=0;i<3;i++) {
                tctpct[i]    =        tp[i] -     c*t[i];
                tpctctp[i]   =         t[i] -    c*tp[i];
                tcbpct[i]    =        bp[i] -  tdbp*t[i];
                tpcbctp[i]   =         b[i] - tpdb*tp[i];
                bpctpct[i]   =   tdbp*tp[i] -    c*bp[i];
                bctctp[i]    =    tpdb*t[i] -     c*b[i];
            }
                
            
            tctpcbpdtp = tdbp - tpdbp*c;
            tpctcbdt = tpdb - tdb*c;
            tctpcbpdb =  tdbp*tpdb - tpdbp*tdb;
            tpctcbdbp = tctpcbpdb;
            tcbpdtp = tpctdbp; 
            tpcbdt = tctpdb;
            tcbpdb = bctdbp;
            tpcbdbp = bpctpdb;

/*
 *          Only calculate the forces for segment p3->p4 if at least one
 *          of the segment's nodes is local to the current domain.
 */
            //if (seg34Local) {

                temp1 = tdbp*tpdb + tctpcbpdb;
                
                #pragma unroll
                for (i=0;i<3;i++) {
                    I00a[i] = temp1 * tpct[i];
                    I00b[i] = tctpcbpdtp * bct[i];
                }

                temp1 = (m4pnd * tctpdb);
                temp2 = (m4pnd * bpctpdb);
                temp3 = (m4pnd3 * tctpcbpdtp*tctpdb);
                
                #pragma unroll
                for (i=0;i<3;i++) {
                    I_003[i] = m4pd*I00a[i] - m4pnd*I00b[i] + temp1*bpctpct[i] +
                            temp2*tctpct[i]; 
                    I_005[i] = a2m8pd*I00a[i] - a2m4pnd*I00b[i] - temp3*tctpct[i];
                    I10a[i] = tcbpct[i]*tpdb - tctp[i]*tcbpdb;
                    I10b[i] = bct[i] * tcbpdtp;
                    
                }

                temp1 = (m4pn * tdb);
                temp2 = m4pnd2 * (tcbpdtp*tctpdb + tctpcbpdtp*tdb);
                
                #pragma unroll
                for (i=0;i<3;i++) {
                    I_103[i] = temp1*bpctpct[i] + m4p*I10a[i] - m4pn*I10b[i];
                    I_105[i] = a2m8p*I10a[i] - a2m4pn*I10b[i] - temp2*tctpct[i];
                    I01a[i] = tctp[i]*bpctpdb - bpctpct[i]*tpdb;
                }

                tmp[0] = (m4pn * tpdb); 
                tmp[1] = (m4pn * bpctpdb);
                tmp[2] = (m4pnd2 * tctpcbpdtp * tpdb);
                tmp[3] = (m4pnd2 * tctpcbpdtp * tctpdb);
                tmp[4] = (m4pnd * tcbpdtp * tdb);
                tmp[5] = (m4pnd * tctpcbpdtp * tpdb) ;
                tmp[6] = (m4pnd * (tctpcbpdtp*tdb + tcbpdtp*tctpdb));
                tmp[7] = (m4pnd * tcbpdtp * tpdb);
                tmp[8] = (m4pn * tcbpdtp * tdb);
                tmp[9] = (m4pn * tcbpdtp * tpdb);
                
                #pragma unroll
                for (i=0;i<3;i++) {
                    I_013[i] = m4p*I01a[i] + tmp[0]*bpctpct[i] - tmp[1]*tctp[i];
                    I_015[i] = a2m8p*I01a[i] - tmp[2]*tctpct[i] + tmp[3]*tctp[i];
                    I_205[i] = -tmp[4] * tctpct[i];
                    I_025[i] = tmp[5] * tctp[i]; 
                    I_115[i] =  tmp[6]*tctp[i] - tmp[7]*tctpct[i];
                    I_215[i] = tmp[8] * tctp[i];
                    I_125[i] = tmp[9] * tctp[i];
                }
  
                Fint_003 = f_103 - y[0]*f_003;
                Fint_103 = f_203 - y[0]*f_103;
                Fint_013 = f_113 - y[0]*f_013;
                Fint_005 = f_105 - y[0]*f_005;
                Fint_105 = f_205 - y[0]*f_105;
                Fint_015 = f_115 - y[0]*f_015;
                Fint_115 = f_215 - y[0]*f_115;
                Fint_205 = f_305 - y[0]*f_205;
                Fint_025 = f_125 - y[0]*f_025;
                Fint_215 = f_315 - y[0]*f_215;
                Fint_125 = f_225 - y[0]*f_125;
                
                #pragma unroll
                for (i=0;i<3;i++) {
                    f4[i]=(I_003[i]*Fint_003 + I_103[i]*Fint_103 + I_013[i]*Fint_013 +
                           I_005[i]*Fint_005 + I_105[i]*Fint_105 + I_015[i]*Fint_015 +
                           I_115[i]*Fint_115 + I_205[i]*Fint_205 + I_025[i]*Fint_025 +
                           I_215[i]*Fint_215 + I_125[i]*Fint_125) * oneoverL;
                }

                Fint_003 = y[1]*f_003 - f_103;
                Fint_103 = y[1]*f_103 - f_203;
                Fint_013 = y[1]*f_013 - f_113;
                Fint_005 = y[1]*f_005 - f_105;
                Fint_105 = y[1]*f_105 - f_205;
                Fint_015 = y[1]*f_015 - f_115;
                Fint_115 = y[1]*f_115 - f_215;
                Fint_205 = y[1]*f_205 - f_305;
                Fint_025 = y[1]*f_025 - f_125;
                Fint_215 = y[1]*f_215 - f_315;
                Fint_125 = y[1]*f_125 - f_225;
                
                #pragma unroll
                for (i=0;i<3;i++) {
                    f3[i]=(I_003[i]*Fint_003 + I_103[i]*Fint_103 + I_013[i]*Fint_013 +
                           I_005[i]*Fint_005 + I_105[i]*Fint_105 + I_015[i]*Fint_015 +
                           I_115[i]*Fint_115 + I_205[i]*Fint_205 + I_025[i]*Fint_025 +
                           I_215[i]*Fint_215 + I_125[i]*Fint_125) * oneoverL;
                }

                fn3.x=f3[0];
                fn3.y=f3[1];
                fn3.z=f3[2];
                fn4.x=f4[0];
                fn4.y=f4[1];
                fn4.z=f4[2];

            //} /* if segment p3->p4 is "local" */

/*
 *          Only calculate the forces for segment p1->p2 if at least one
 *          of the segment's nodes is local to the current domain.
 */
            //if (seg12Local) {

                temp1 = tpdb*tdbp + tpctcbdbp;
                
                #pragma unroll
                for (i=0;i<3;i++) {
                    I00a[i] = temp1 * tctp[i];
                    I00b[i] = bpctp[i] * tpctcbdt;
                }
                
                temp1 = m4pnd * tpctdbp;
                temp2 = m4pnd * bctdbp;
                temp3 = m4pnd3 * tpctcbdt * tpctdbp;
                
                #pragma unroll
                for (i=0;i<3;i++) {
                    I_003[i] = m4pd*I00a[i] - m4pnd*I00b[i] + temp1*bctctp[i] +
                               temp2*tpctctp[i];
                    I_005[i] = a2m8pd*I00a[i] - a2m4pnd*I00b[i] - temp3*tpctctp[i]; 
                    I01a[i] = tpct[i]*tpcbdbp - tpcbctp[i]*tdbp;
                    I01b[i] = -bpctp[i] * tpcbdt;
                }

                temp1 = m4pn * tpdbp;
                temp2 = m4pnd2 * (tpcbdt*tpctdbp + tpctcbdt*tpdbp);
                
                #pragma unroll
                for (i=0;i<3;i++) {
                    I_013[i] = -temp1 * bctctp[i] + m4p*I01a[i] - m4pn*I01b[i];
                    I_015[i] = a2m8p*I01a[i] - a2m4pn*I01b[i] + temp2*tpctctp[i];
                    I10a[i] = bctctp[i]*tdbp - tpct[i]*bctdbp;
                }

                tmp[0] = m4pn * tdbp; 
                tmp[1] = m4pn * bctdbp;
                tmp[2] = m4pnd2 * tpctcbdt * tdbp;
                tmp[3] = m4pnd2 * tpctcbdt * tpctdbp;
                tmp[4] = (m4pnd * tpcbdt * tpdbp);
                tmp[5] = (m4pnd * tpctcbdt * tdbp);
                tmp[6] = m4pnd * (tpctcbdt*tpdbp + tpcbdt*tpctdbp);
                tmp[7] = m4pnd * tpcbdt * tdbp;
                tmp[8] = (m4pn * tpcbdt * tpdbp);
                tmp[9] = (m4pn * tpcbdt * tdbp);
                
                #pragma unroll
                for (i=0;i<3;i++) {
                    I_103[i] = m4p*I10a[i] - tmp[0]*bctctp[i] + tmp[1]*tpct[i];
                    I_105[i] = a2m8p*I10a[i] + tmp[2]*tpctctp[i] - tmp[3]*tpct[i];
                    I_025[i] = -tmp[4] * tpctctp[i];
                    I_205[i] = tmp[5] * tpct[i];
                    I_115[i] = tmp[6]*tpct[i] - tmp[7]*tpctctp[i];
                    I_125[i] = -tmp[8] * tpct[i];
                    I_215[i] = -tmp[9] * tpct[i];
                }

                Fint_003 = f_013 - z[1]*f_003;
                Fint_103 = f_113 - z[1]*f_103;
                Fint_013 = f_023 - z[1]*f_013;
                Fint_005 = f_015 - z[1]*f_005;
                Fint_105 = f_115 - z[1]*f_105;
                Fint_015 = f_025 - z[1]*f_015;
                Fint_115 = f_125 - z[1]*f_115;
                Fint_205 = f_215 - z[1]*f_205;
                Fint_025 = f_035 - z[1]*f_025;
                Fint_215 = f_225 - z[1]*f_215;
                Fint_125 = f_135 - z[1]*f_125;
                
                #pragma unroll
                for (i=0;i<3;i++) {
                    f1[i]=(I_003[i]*Fint_003 + I_103[i]*Fint_103 + I_013[i]*Fint_013 +
                           I_005[i]*Fint_005 + I_105[i]*Fint_105 + I_015[i]*Fint_015 +
                           I_115[i]*Fint_115 + I_205[i]*Fint_205 + I_025[i]*Fint_025 +
                           I_215[i]*Fint_215 + I_125[i]*Fint_125) * oneoverLp;
                }
   
                Fint_003 = z[0]*f_003 - f_013;
                Fint_103 = z[0]*f_103 - f_113;
                Fint_013 = z[0]*f_013 - f_023;
                Fint_005 = z[0]*f_005 - f_015;
                Fint_105 = z[0]*f_105 - f_115;
                Fint_015 = z[0]*f_015 - f_025;
                Fint_115 = z[0]*f_115 - f_125;
                Fint_205 = z[0]*f_205 - f_215;
                Fint_025 = z[0]*f_025 - f_035;
                Fint_215 = z[0]*f_215 - f_225;
                Fint_125 = z[0]*f_125 - f_135;
                
                #pragma unroll
                for (i=0;i<3;i++) {
                    f2[i]=(I_003[i]*Fint_003 + I_103[i]*Fint_103 + I_013[i]*Fint_013 +
                           I_005[i]*Fint_005 + I_105[i]*Fint_105 + I_015[i]*Fint_015 +
                           I_115[i]*Fint_115 + I_205[i]*Fint_205 + I_025[i]*Fint_025 +
                           I_215[i]*Fint_215 + I_125[i]*Fint_125) * oneoverLp;
                }
                
                fn1.x=f1[0];
                fn1.y=f1[1];
                fn1.z=f1[2];
                fn2.x=f2[0];
                fn2.y=f2[1];
                fn2.z=f2[2];
                
   
            //} /* if segment p1->p2 is "local" */

        } else {
/*
 *          The two lines are parallel, so we have to use a special
 *          lower dimensional function
 */
			
			SpecialSegSegForceHalfGPU(r1, r2, r3, r4, b1, b2, fn3, fn4);

            SpecialSegSegForceHalfGPU(r3, r4, r1, r2, b2, b1, fn1, fn2);
       }

       return;
}

/*------------------------------------------------------------------------
 *
 *      Function:    DotProductGPU
 *
 *-----------------------------------------------------------------------*/
__device__ double DotProductGPU(double3 v1, double3 v2)
{
	return v1.x*v2.x + v1.y*v2.y + v1.z*v2.z;
}

/*------------------------------------------------------------------------
 *
 *      Function:    ZImageGPU
 *
 *-----------------------------------------------------------------------*/
__device__ void ZImageGPU(double3 &r)
{
/*
 *      If periodic boundaries are not in use, the provided position
 *      of (x,y,z) will not be adjusted since there are no other
 *      images available (in that case Lx,Ly,Lz = 0).
 */
		r.x -= rint(r.x * invLx) * Lx;
		r.y -= rint(r.y * invLy) * Ly;
		r.z -= rint(r.z * invLz) * Lz;
}

/*------------------------------------------------------------------------
 *
 *      Function:    FoldBoxGPU
 *
 *-----------------------------------------------------------------------*/
__device__ void FoldBoxGPU(double3 &r)
{
		r.x -= rint((r.x-boxc.x)*invLx) * Lx;
		r.y -= rint((r.y-boxc.y)*invLy) * Ly;
		r.z -= rint((r.z-boxc.z)*invLz) * Lz;
}

/*------------------------------------------------------------------------
 *
 *      Function:    PreserveNodesGPU
 *
 *-----------------------------------------------------------------------*/
__global__ void PreserveNodesGPU(int nodeCount, double3 *r, double3 *r0, double3 *v, double3 *v0)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < nodeCount) {
		r0[i] = r[i];
		v0[i] = v[i];
	}
}

/*------------------------------------------------------------------------
 *
 *      Function:    ResetNodesGPU
 *
 *-----------------------------------------------------------------------*/
__global__ void ResetNodesGPU(int nodeCount, double3 *v, double3 *v0)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < nodeCount) {
		v[i] = v0[i];
	}
}

/*------------------------------------------------------------------------
 *
 *      Function:    ResetForceGPU
 *
 *-----------------------------------------------------------------------*/
__global__ void ResetForceGPU(int nodeCount, double3 *f)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < nodeCount) {
		f[i].x = 0.0;
		f[i].y = 0.0;
		f[i].z = 0.0;
	}
}

/*------------------------------------------------------------------------
 *
 *      Function:    MobilityDragGPU
 *
 *-----------------------------------------------------------------------*/
__device__ void MobilityDragGPU(double3 dr, double L, double3 b, double &B)
{
	double3 ub;
	double  magb, dangle, Mob;
	
	if (L < 1.e-10) {
		B = 1.0; // avoid division-by-0 later
		return;
	}
	
	magb = sqrt(b.x*b.x + b.y*b.y + b.z*b.z);
	ub.x = b.x / magb;
	ub.y = b.y / magb;
	ub.z = b.z / magb;
	
	dangle = fabs(ub.x*dr.x + ub.y*dr.y + ub.z*dr.z);
	Mob = MobEdge+(MobScrew-MobEdge)*dangle;
	B = 0.5*L/Mob;
}

/*------------------------------------------------------------------------
 *
 *      Function:    SelfForceGPU
 *
 *-----------------------------------------------------------------------*/
template <unsigned int coreOnly>
__device__ void SelfForceGPU(double3 dr, double L, double3 b, double3 &f1, double3 &f2)
{
	double3 fs;
	double  Ecore, eps;

	if (coreOnly) {
		Ecore = 0.5 * TensionFactor * MU;
		eps = 1.e-06;
	} else {
		Ecore = coreEnergy;
		eps = 1.e-20;
	}
	
	if (L*L < eps) {
		return;
	}

	double tx, ty, tz, La, S;
	double bs, bs2, bex, bey, bez, be2, fL, ft;

	tx = dr.x / L;
	ty = dr.y / L;
	tz = dr.z / L;

	bs = b.x*tx + b.y*ty + b.z*tz;
	bex = b.x-bs*tx; bey = b.y-bs*ty; bez=b.z-bs*tz;
	be2 = (bex*bex+bey*bey+bez*bez);
	bs2 = bs*bs;

	La = sqrt(L*L+a*a);

	if (coreOnly) {
		S = 0.0;
	} else {
		S = (-(2*NU*La+(1-NU)*a*a/La-(1+NU)*a)/L +
			 (NU*log((La+L)/a)-(1-NU)*0.5*L/La))*MU/4/M_PI/(1-NU)*bs;
	}

	/* Ecore = MU/(4*pi) log(a/a0) */
	fL = -Ecore*(bs2+be2/(1-NU));
	ft =  Ecore*2*bs*NU/(1-NU); 

	fs.x = bex*(S+ft) + fL*tx;
	fs.y = bey*(S+ft) + fL*ty;
	fs.z = bez*(S+ft) + fL*tz;

	f2.x += fs.x;
	f2.y += fs.y;
	f2.z += fs.z;

	f1.x -= fs.x;
	f1.y -= fs.y;
	f1.z -= fs.z;
}

/*------------------------------------------------------------------------
 *
 *      Function:    ExtForceGPU
 *
 *-----------------------------------------------------------------------*/
__device__ void ExtForceGPU(double3 dr, double3 b, double *extStress, double3 &f1, double3 &f2)
{
	double3 sigb, ft;
	
	sigb.x = extStress[0]*b.x + extStress[5]*b.y + extStress[4]*b.z;
	sigb.y = extStress[5]*b.x + extStress[1]*b.y + extStress[3]*b.z;
	sigb.z = extStress[4]*b.x + extStress[3]*b.y + extStress[2]*b.z;
	
	ft.x = sigb.y*dr.z - sigb.z*dr.y;
	ft.y = sigb.z*dr.x - sigb.x*dr.z;
	ft.z = sigb.x*dr.y - sigb.y*dr.x;
	
	f1.x += 0.5*ft.x;
	f1.y += 0.5*ft.y;
	f1.z += 0.5*ft.z;
		
	f2.x += 0.5*ft.x;
	f2.y += 0.5*ft.y;
	f2.z += 0.5*ft.z;
}

/*---------------------------------------------------------------------------
 *
 *      Function:     RemForceGPU
 *
 *-------------------------------------------------------------------------*/
__device__ void RemForceGPU(double3 dr, double3 b, double3 *sigb, double3 &f1, double3 &f2)
{
	int       i, numPoints;
	double    positions[3], weights[3];
	double    temp, mult1, mult2;
	double    sigbx, sigby, sigbz;
	double    fLinvx, fLinvy, fLinvz;
	double    pspanx, pspany, pspanz;
	
	numPoints = 3;
	positions[0] = -0.774596669241483;
	positions[1] = 0.0;
	positions[2] = -positions[0];
	weights[0] = 0.5*5.0/9.0;
	weights[1] = 0.5*8.0/9.0;
	weights[2] = weights[0];
	
	pspanx = 0.5 * dr.x;
	pspany = 0.5 * dr.y;
	pspanz = 0.5 * dr.z;
	
	for (i = 0; i < numPoints; i++) {
		
		sigbx = sigb[i].x;
		sigby = sigb[i].y;
		sigbz = sigb[i].z;

		fLinvx = (sigby*pspanz-sigbz*pspany);
		fLinvy = (sigbz*pspanx-sigbx*pspanz);
		fLinvz = (sigbx*pspany-sigby*pspanx);

		temp = weights[i]*positions[i];
		mult1 = weights[i]+temp;

		f2.x += fLinvx*mult1;
		f2.y += fLinvy*mult1;
		f2.z += fLinvz*mult1;

		mult2 = weights[i]-temp;

		f1.x += fLinvx*mult2;
		f1.y += fLinvy*mult2;
		f1.z += fLinvz*mult2;    
	}
}

/*------------------------------------------------------------------------
 *
 *      Function:    SegForceDragGPU
 *
 *-----------------------------------------------------------------------*/
template <unsigned int elasticinteraction>
__global__ void SegForceDragGPU(int segCount, double3 *r, int2 *s, double3 *b, double *extStress, double3 *fseg, double *Bseg, double3 *fmm)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < segCount) {
		
		int2 n = s[i];

		double3 r1, r2, dr, bs, f1, f2;
		double  L, B;

		r1 = r[n.x];
		r2 = r[n.y];
		bs = b[i];
		
		dr.x = r2.x - r1.x;
		dr.y = r2.y - r1.y;
		dr.z = r2.z - r1.z;
		ZImageGPU(dr);
		L = sqrt(dr.x*dr.x + dr.y*dr.y + dr.z*dr.z);

		f1.x = 0.0;
		f1.y = 0.0;
		f1.z = 0.0;

		f2.x = 0.0;
		f2.y = 0.0;
		f2.z = 0.0;

		switch(elasticinteraction) {
			case 0:
				SelfForceGPU<1>(dr, L, bs, f1, f2);
				ExtForceGPU(dr, bs, extStress, f1, f2);
				break;
			case 1:
				SelfForceGPU<0>(dr, L, bs, f1, f2);
				ExtForceGPU(dr, bs, extStress, f1, f2);
				
				int numPoints = 3;
				double3 sigb[3];
				sigb[0] = fmm[i*numPoints+0];
				sigb[1] = fmm[i*numPoints+1];
				sigb[2] = fmm[i*numPoints+2];
				
				double3 fs1, fs2;
				fs1.x = 0.0; fs1.y = 0.0; fs1.z = 0.0;
				fs2.x = 0.0; fs2.y = 0.0; fs2.z = 0.0;
				
				RemForceGPU(dr, bs, sigb, fs1, fs2);
				
				f1.x += fs1.x;
				f1.y += fs1.y;
				f1.z += fs1.z;
				
				f2.x += fs2.x;
				f2.y += fs2.y;
				f2.z += fs2.z;
				break;
		}

		fseg[i*2+0] = f1;
		fseg[i*2+1] = f2;

		MobilityDragGPU(dr, L, bs, B);
		Bseg[i] = B;
	}
}

/*------------------------------------------------------------------------
 *
 *      Function:    SegDragGPU
 *
 *-----------------------------------------------------------------------*/
__global__ void SegDragGPU(int segCount, double3 *r, int2 *s, double3 *b, double *Bseg)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < segCount) {
		
		int2 n = s[i];

		double3 r1, r2, dr, bs;
		double  L, B;

		r1 = r[n.x];
		r2 = r[n.y];
		bs = b[i];
		
		dr.x = r2.x - r1.x;
		dr.y = r2.y - r1.y;
		dr.z = r2.z - r1.z;
		ZImageGPU(dr);
		L = sqrt(dr.x*dr.x + dr.y*dr.y + dr.z*dr.z);

		MobilityDragGPU(dr, L, bs, B);
		Bseg[i] = B;
	}
}

/*------------------------------------------------------------------------
 *
 *      Function:    SegSegForceGPU
 *
 *-----------------------------------------------------------------------*/
template <unsigned int checkFlags>
__global__ void SegSegForceGPU(int segSegCount, double3 *r, int2 *s, double3 *b, int2 *g, double3 *cc, int *gflag, double3 *fseg)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < segSegCount) {
		
		int2 segs = g[i];
		
		int2 s1 = s[segs.x];
		int2 s2 = s[segs.y];
		
		double3 b1, b2;
		b1 = b[segs.x];
		b2 = b[segs.y];
		
		double3 r1, r2, r3, r4;
		double3 f1, f2, f3, f4;
		
		int n1 = s1.x;
		int n2 = s1.y;
		int n3 = s2.x;
		int n4 = s2.y;
		
		r1 = r[n1];
		r2 = r[n2];
		r3 = r[n3];
		r4 = r[n4];
		
		int compute = 1;
		if (checkFlags == 1) {
			compute = gflag[i];
			if (compute == 0) compute = 1;
			else compute = 0;
		}
		
		double3 dr1, dr2;
		double  L1s, L2s;
		dr1.x = r2.x - r1.x;
		dr1.y = r2.y - r1.y;
		dr1.z = r2.z - r1.z;
		ZImageGPU(dr1);
		L1s = dr1.x*dr1.x + dr1.y*dr1.y + dr1.z*dr1.z;
		
		dr2.x = r4.x - r3.x;
		dr2.y = r4.y - r3.y;
		dr2.z = r4.z - r3.z;
		ZImageGPU(dr2);
		L2s = dr2.x*dr2.x + dr2.y*dr2.y + dr2.z*dr2.z;
		
		if (L1s < 1.e-20 || L2s < 1.e-20 || !compute) {
			
			f1.x = 0.0; f1.y = 0.0; f1.z = 0.0;
			f2.x = 0.0; f2.y = 0.0; f2.z = 0.0;
			f3.x = 0.0; f3.y = 0.0; f3.z = 0.0;
			f4.x = 0.0; f4.y = 0.0; f4.z = 0.0;
			
		} else {
		
			r2.x = r1.x + dr1.x;
			r2.y = r1.y + dr1.y;
			r2.z = r1.z + dr1.z;
			
			// Cell center here??
			double3 rc;
			//rc = r1;
			rc = cc[n1];
			dr1.x = r3.x - rc.x;
			dr1.y = r3.y - rc.y;
			dr1.z = r3.z - rc.z;
			ZImageGPU(dr1);
			
			r3.x = rc.x + dr1.x;
			r3.y = rc.y + dr1.y;
			r3.z = rc.z + dr1.z;
			
			dr2.x = r4.x - r3.x;
			dr2.y = r4.y - r3.y;
			dr2.z = r4.z - r3.z;
			ZImageGPU(dr2);
			
			r4.x = r3.x + dr2.x;
			r4.y = r3.y + dr2.y;
			r4.z = r3.z + dr2.z;
			
			SegSegForceIsotropicGPU(r1, r2, r3, r4, b1, b2, f1, f2, f3, f4);
			
		}
		
		fseg[i*4+0] = f1;
		fseg[i*4+1] = f2;
		fseg[i*4+2] = f3;
		fseg[i*4+3] = f4;
	}
	
	return;
}

/*------------------------------------------------------------------------
 *
 *      Function:    ReduceForceGPU
 *
 *-----------------------------------------------------------------------*/
template <unsigned int incrementForces>
__global__ void ReduceForceGPU(int nodeCount, int2 *gpos, int *gind, double3 *fseg, double3 *f)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < nodeCount) {
		
		int2 pos = gpos[i];
		int j, k = 0;
		
		double3 fs, ft;
		ft.x = 0.0;
		ft.y = 0.0;
		ft.z = 0.0;
		
		for (k = pos.x; k < pos.y; k++) {
			j = gind[k];
			fs = fseg[j];
			ft.x += fs.x;
			ft.y += fs.y;
			ft.z += fs.z;
		}
		
		if (incrementForces == 0) {
			f[i] = ft;
		} else {
			f[i].x += ft.x;
			f[i].y += ft.y;
			f[i].z += ft.z;
		}
	}
}

/*------------------------------------------------------------------------
 *
 *      Function:    ResetArmForceGPU
 *
 *-----------------------------------------------------------------------*/
__global__ void ResetArmForceGPU(int armCount, double3 *farms)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < armCount) {
		farms[i].x = 0.0;
		farms[i].y = 0.0;
		farms[i].z = 0.0;
	}
}

/*------------------------------------------------------------------------
 *
 *      Function:    ReduceArmForceGPU
 *
 *-----------------------------------------------------------------------*/
__global__ void ReduceArmForceGPU(int armCount, int *garms, double3 *fseg, double3 *farms)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < armCount) {
		
		int pos = garms[i];
		
		double3 fs;
		fs = fseg[pos];
		
		farms[i].x += fs.x;
		farms[i].y += fs.y;
		farms[i].z += fs.z;
	}
}

/*------------------------------------------------------------------------
 *
 *      Function:    ReduceArmForceGPU
 *
 *-----------------------------------------------------------------------*/
__global__ void ReduceArmForceGPU(int armCount, int2 *garms_pos, int *garms_ind, double3 *fseg, double3 *farms)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < armCount) {
		
		int2 pos = garms_pos[i];
		int j, k;
		
		double3 fs, ft;
		ft.x = 0.0;
		ft.y = 0.0;
		ft.z = 0.0;
		
		for (k = pos.x; k < pos.y; k++) {
			j = garms_ind[k];
			fs = fseg[j];
			ft.x += fs.x;
			ft.y += fs.y;
			ft.z += fs.z;
		}
		
		farms[i].x += ft.x;
		farms[i].y += ft.y;
		farms[i].z += ft.z;
	}
}

/*------------------------------------------------------------------------
 *
 *      Function:    ReduceDragGPU
 *
 *-----------------------------------------------------------------------*/
__global__ void ReduceDragGPU(int nodeCount, int2 *gpos, int *gind, double *Bseg, double *B)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < nodeCount) {
		
		int2 pos = gpos[i];
		int j, k = 0;
		
		double Bs, Bt;
		Bt = 0.0;
		
		for (k = pos.x; k < pos.y; k++) {
			j = gind[k]/2;
			Bs = Bseg[j];
			Bt += Bs;
		}
		B[i] = Bt;
	}
}

/*------------------------------------------------------------------------
 *
 *      Function:    NodeVelocityGPU
 *
 *-----------------------------------------------------------------------*/
__global__ void NodeVelocityGPU(int nodeCount, double3 *f, double *B, double *mob, int *n, double3 *v)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < nodeCount) {
		
		double3 fn, vn;
		double Bn, mobn[9];
		
		fn = f[i];
		Bn = B[i];
		if (Bn == 0.0) Bn = 1.0; // avoid division-by-zero
		
		for (int j = 0; j < 9; j++) {
			mobn[j] = mob[i*9+j];
		}
		
		fn.x /= Bn;
		fn.y /= Bn;
		fn.z /= Bn;
		
		// Project velocity onto glide constraints
		vn.x = mobn[0] * fn.x + mobn[1] * fn.y + mobn[2] * fn.z;
		vn.y = mobn[3] * fn.x + mobn[4] * fn.y + mobn[5] * fn.z;
		vn.z = mobn[6] * fn.x + mobn[7] * fn.y + mobn[8] * fn.z;

		// Oscillating node
		int a = n[i];
		if (a == 0) {
			vn.x = 0.0;
			vn.y = 0.0;
			vn.z = 0.0;
		}

		v[i] = vn;
	}
}

/*------------------------------------------------------------------------
 *
 *      Function:    RKFStepGPU
 *
 *-----------------------------------------------------------------------*/
//template <unsigned int step>
template <unsigned int moveNodes, unsigned int blockSize>
__global__ void RKFStepGPU(int step, int nodeCount, double currDT, double3 *v, double3 *rkf, double3 *r0, double3 *r, double *e1, double *e2, double3 *f, int *n)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int jmax = step+1;
	double3 vs, rrkf[6];
	double3 rold, rnew, rcur;
	
	if (i < nodeCount) {

		vs = v[i];
		rrkf[step] = vs;
		if (step < 5) rkf[i*5+step] = vs;
		for (int j = 0; j < step; j++) {
			rrkf[j] = rkf[i*5+j];
		}

		double c[6];
		if (step == 0) {
			c[0] = 1.0/4.0;
		}
		if (step == 1) {
			c[0] = 3.0/32.0;
			c[1] = 9.0/32.0;
		}
		if (step == 2) {
			c[0] = 1932.0/2197.0;
			c[1] = -7200.0/2197.0;
			c[2] = 7296.0/2197.0;
		}
		if (step == 3) {
			c[0] = 439.0/216.0;
			c[1] = -8.0;
			c[2] = 3680.0/513.0;
			c[3] = -845.0/4104.0;
		}
		if (step == 4) {
			c[0] = -8.0/27.0;
			c[1] = 2.0;
			c[2] = -3544.0/2565.0;
			c[3] = 1859.0/4104.0;
			c[4] = -11.0/40.0;
		}
		if (step == 5) {
			c[0] = 16.0/135.0;
			c[1] = 0.0;
			c[2] = 6656.0/12825.0;
			c[3] = 28561.0/56430.0;
			c[4] = -9.0/50.0;
			c[5] = 2.0/55.0;
		}

		rold = r0[i];
		rnew.x = 0.0;
		rnew.y = 0.0;
		rnew.z = 0.0;
		for (int j = 0; j < jmax; j++) {
			rnew.x += c[j]*rrkf[j].x;
			rnew.y += c[j]*rrkf[j].y;
			rnew.z += c[j]*rrkf[j].z;
		}
		rnew.x = rold.x + currDT*rnew.x;
		rnew.y = rold.y + currDT*rnew.y;
		rnew.z = rold.z + currDT*rnew.z;
		
		FoldBoxGPU(rnew);
		
		if (step == 5) {
			rcur = r[i]; // WARNING
		}
		//rcur = rnew;
		
		r[i] = rnew;
	}

	// Error calculation
	if (step == 5) {
		
		double er[6];
		er[0] =  1.0/360.0;
		er[1] =  0.0;
		er[2] = -128.0/4275.0;
		er[3] = -2197.0/75240.0;
		er[4] =  1.0/50.0;
		er[5] =  2.0/55.0;

		double errnet = 0.0;
		double relerrnet = 0.0;
		if (i < nodeCount) {
			
			double3 err;
			err.x = 0.0;
			err.y = 0.0;
			err.z = 0.0;
			for (int j = 0; j < jmax; j++) {
				err.x += er[j]*rrkf[j].x;
				err.y += er[j]*rrkf[j].y;
				err.z += er[j]*rrkf[j].z;
			}
			err.x *= currDT;
			err.y *= currDT;
			err.z *= currDT;
			errnet = sqrt(err.x*err.x+err.y*err.y+err.z*err.z); //sqrtf
			
			double3 dr;
			dr.x = rcur.x - rold.x;
			dr.y = rcur.y - rold.y;
			dr.z = rcur.z - rold.z;
			ZImageGPU(dr);
			
			double drn;
			drn = sqrt(dr.x*dr.x + dr.y*dr.y + dr.z*dr.z);
			if (errnet > rTolth) {
				if (drn > rTolth/rTolrel) {
					relerrnet = errnet / drn;
				} else {
					relerrnet = 2*rTolrel;
				}
			}
			
			if (moveNodes == 1) {
				if (errnet < rTol && (errnet < rTolth || errnet/drn < rTolrel)) {
					n[i] = 1; // unflag node
				} else {
					n[i] = 2; // flag node
				}
			}
		}

		// Reduce block error
		__shared__ double errtmp1[blockSize];
		__shared__ double errtmp2[blockSize];
		
		int tid = threadIdx.x;
		errtmp1[tid] = errnet;
		errtmp2[tid] = relerrnet;
		__syncthreads();

		for (unsigned int s = blockDim.x/2; s >= 1; s = s/2) {
			if (tid < s) {
				if (errtmp1[tid] < errtmp1[tid + s])
					errtmp1[tid] = errtmp1[tid + s];
				if (errtmp2[tid] < errtmp2[tid + s])
					errtmp2[tid] = errtmp2[tid + s];
			}
			__syncthreads();
		}

		if (tid == 0) {
			e1[blockIdx.x] = errtmp1[0];
			e2[blockIdx.x] = errtmp2[0];
		}
	}
}

/*------------------------------------------------------------------------
 *
 *      Function:    FlagNodesGPU
 *
 *-----------------------------------------------------------------------*/
__global__ void FlagNodesGPU(int nodeCount, int *n)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < nodeCount) {
		n[i] = 1;
	}
}

/*------------------------------------------------------------------------
 *
 *      Function:    MoveInteractionsGPU
 *
 *-----------------------------------------------------------------------*/
__global__ void MoveInteractionsGPU(int segSegCount, double rgs, int2 *s, int2 *g, double *gdist2, int *n, int *gflag)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < segSegCount) {
		
		int2 segs = g[i];
		int2 s1 = s[segs.x];
		int2 s2 = s[segs.y];
		
		int n1 = s1.x;
		int n2 = s1.y;
		int n3 = s2.x;
		int n4 = s2.y;
		
		int a1 = n[n1];
		int a2 = n[n2];
		int a3 = n[n3];
		int a4 = n[n4];
		
		// Flag the interaction to be moved to group 4 
		// if any of its nodes is flagged 2
		if ((a1-2)*(a2-2)*(a3-2)*(a4-2) == 0) {
			if (gdist2[i] <= rgs && gflag[i] == 0) {
				gflag[i] = 4;
			}
		}
	}
}

/*------------------------------------------------------------------------
 *
 *      Function:    ForwardProgressCheckGPU
 *
 *-----------------------------------------------------------------------*/
__global__ void ForwardProgressCheckGPU(int nodeCount, double3 *v0, double3 *v, int *n)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < nodeCount) {
		double3 vold, vnew;
		vold = v0[i];
		vnew = v[i];
		
		double vv;
		vv = vold.x*vnew.x + vold.y*vnew.y + vold.z*vnew.z;
		if (vv < 0.0) n[i] = 0;
	}
}

/*------------------------------------------------------------------------
 *
 *      Function:    RKFIntegratorGPU
 *
 *-----------------------------------------------------------------------*/
void RKFIntegratorGPU(Home_t *home, Device_t *device, int reqType, int nSubcyc)
{
		int     i, threadsPerBlock;
		int     blocksNodes, blocksSegs, blocksSegSegs, blocksArms; 
		int     nodeCount, segCount, segSegCount, armCount;
		double  errMax, relErrMax, newDT, rg9s, tmp_rg9s;
		int2    *seg;
		int2    *group;
		int2    *spos, *gpos;
		int     *gind, *sind, *gflag;
		double3 *fseg;
		double3 *fmm;
		Param_t *param;
		
		param = home->param;
		
		
		/* Segment list */
		seg  = device->g1;
		spos = device->g1pos;
		sind = device->g1ind;
		segCount   = device->segCount;
		blocksSegs = device->blocksSegs;
		armCount   = device->armCount;
		blocksArms = device->blocksArms;
		threadsPerBlock = device->threadsPerBlock;
		
		fmm = NULL;
		gflag = NULL;
		
		if (reqType == FULL) {
			
			nodeCount   = device->nodeCount;
			blocksNodes = device->blocksNodes;
			
			fseg = device->fseg;
			gpos = device->g1pos;
			gind = device->g1ind;
			
			if (param->elasticinteraction) {
				fmm  = device->fmm;
			}
			
			if (nSubcyc == -1) {
				gflag = device->g0flag;
			}
			
		} else if (reqType == GROUP0) {
			
			nodeCount     = device->nodeCount;
			blocksNodes   = device->blocksNodes;
			segSegCount   = device->nSegSeg0;
			blocksSegSegs = device->blocksSegSegs0;
			
			fseg  = device->f0;
			group = device->g0;
			gpos  = device->g0pos;
			gind  = device->g0ind;
			gflag = device->g0flag;
			
			rg9s = MAX(MAX(MAX(param->rg1,param->rg2),param->rg3),param->rg4) * 2;
			rg9s = rg9s * rg9s;
			rg9s = 5000.0 *5000.0 ; // WARNING
			//This should probably not be set to a constant ????
			
		} else if (reqType == GROUP1) {
			
			nodeCount   = device->nodeCount;
			blocksNodes = device->blocksNodes;
			
			fseg = device->fseg;
			gpos = device->g1pos;
			gind = device->g1ind;
			
			if (param->forceCutOff == 0 && param->fmEnabled) {
				fmm  = device->fmm;
			}
			
		} else if (reqType == GROUP2) {
			
			nodeCount     = device->nodeCount;
			blocksNodes   = device->blocksNodes;
			segSegCount   = device->nSegSeg2;
			blocksSegSegs = device->blocksSegSegs2;
			
			fseg  = device->f2;
			group = device->g2;
			gpos  = device->g2pos;
			gind  = device->g2ind;
			
		} else if (reqType == GROUP3) {
			
			nodeCount     = device->nodeCount;
			blocksNodes   = device->blocksNodes;
			segSegCount   = device->nSegSeg3;
			blocksSegSegs = device->blocksSegSegs3;
			
			fseg  = device->f3;
			group = device->g3;
			gpos  = device->g3pos;
			gind  = device->g3ind;
			
		} else if (reqType == GROUP4) {
			
			nodeCount     = device->nodeCount;
			blocksNodes   = device->blocksNodes;
			segSegCount   = device->nSegSeg4;
			blocksSegSegs = device->blocksSegSegs4;
			
			fseg  = device->f4;
			group = device->g4;
			gpos  = device->g4pos;
			gind  = device->g4ind;
			
		} else {
			Fatal("GPU subcycling is not available for this subGroup yet!");
		}
		
		
		/* Flag all nodes for subcycling */
		if (nSubcyc <= 0) {
			FlagNodesGPU<<<blocksNodes,threadsPerBlock>>>(nodeCount, device->n);
		}
		
		switch (param->elasticinteraction) {
			case 0:
				if (reqType == FULL || reqType == GROUP1) {
					SegForceDragGPU<0><<<blocksSegs,threadsPerBlock>>>(segCount, device->r, seg, device->b, device->esig, fseg, device->Bseg, fmm);
				} else {
					/* Dummy, just to avoid unitialized memory read later */
					SegDragGPU<<<blocksSegs,threadsPerBlock>>>(segCount, device->r, seg, device->b, device->Bseg);
				}
				break;
			case 1:
				if (reqType == FULL || reqType == GROUP1) {
					SegForceDragGPU<1><<<blocksSegs,threadsPerBlock>>>(segCount, device->r, seg, device->b, device->esig, fseg, device->Bseg, fmm);
					if (reqType == FULL) {
						if (nSubcyc == -1) {
							SegSegForceGPU<1><<<device->blocksSegSegs0,threadsPerBlock>>>(device->nSegSeg0, device->r, seg, device->b, device->g0, device->cc, gflag, device->f0);
						} else {
							SegSegForceGPU<0><<<device->blocksSegSegs0,threadsPerBlock>>>(device->nSegSeg0, device->r, seg, device->b, device->g0, device->cc, gflag, device->f0);
						}
						SegSegForceGPU<0><<<device->blocksSegSegs2,threadsPerBlock>>>(device->nSegSeg2, device->r, seg, device->b, device->g2, device->cc, gflag, device->f2);
						SegSegForceGPU<0><<<device->blocksSegSegs3,threadsPerBlock>>>(device->nSegSeg3, device->r, seg, device->b, device->g3, device->cc, gflag, device->f3);
						SegSegForceGPU<0><<<device->blocksSegSegs4,threadsPerBlock>>>(device->nSegSeg4, device->r, seg, device->b, device->g4, device->cc, gflag, device->f4);
					}
				} else {
					if (reqType == GROUP0) {
						SegSegForceGPU<1><<<blocksSegSegs,threadsPerBlock>>>(segSegCount, device->r, seg, device->b, group, device->cc, gflag, fseg);
					} else {
						SegSegForceGPU<0><<<blocksSegSegs,threadsPerBlock>>>(segSegCount, device->r, seg, device->b, group, device->cc, gflag, fseg);
					}
					SegDragGPU<<<blocksSegs,threadsPerBlock>>>(segCount, device->r, seg, device->b, device->Bseg);
				}
				break;
		}
		
		if (reqType == FULL) {
			ResetForceGPU<<<blocksNodes,threadsPerBlock>>>(nodeCount, device->f);
			ReduceForceGPU<1><<<blocksNodes,threadsPerBlock>>>(nodeCount, device->g0pos, device->g0ind, device->f0, device->f);
			ReduceForceGPU<1><<<blocksNodes,threadsPerBlock>>>(nodeCount, device->g1pos, device->g1ind, device->fseg, device->f);
			ReduceForceGPU<1><<<blocksNodes,threadsPerBlock>>>(nodeCount, device->g2pos, device->g2ind, device->f2, device->f);
			ReduceForceGPU<1><<<blocksNodes,threadsPerBlock>>>(nodeCount, device->g3pos, device->g3ind, device->f3, device->f);
			ReduceForceGPU<1><<<blocksNodes,threadsPerBlock>>>(nodeCount, device->g4pos, device->g4ind, device->f4, device->f);
		} else {
			ReduceForceGPU<0><<<blocksNodes,threadsPerBlock>>>(nodeCount, gpos, gind, fseg, device->f);
		}
		ReduceDragGPU<<<blocksNodes,threadsPerBlock>>>(nodeCount, spos, sind, device->Bseg, device->B);
		
		NodeVelocityGPU<<<blocksNodes,threadsPerBlock>>>(nodeCount, device->f, device->B, device->mob, device->n, device->v);
		PreserveNodesGPU<<<blocksNodes,threadsPerBlock>>>(nodeCount, device->r, device->r0, device->v, device->v0);
		
		
		/* Force velocity calculation only */
		if (nSubcyc == -1) {
			if (reqType == FULL) {
				/* When we are doing a full calculation, we need to communicate the arm forces back as well */
				ResetArmForceGPU<<<blocksArms,threadsPerBlock>>>(armCount, device->farms);
				ReduceArmForceGPU<<<blocksArms,threadsPerBlock>>>(armCount, device->g0arms_pos, device->g0arms_ind, device->f0, device->farms);
				ReduceArmForceGPU<<<blocksArms,threadsPerBlock>>>(armCount, device->g1arms, device->fseg, device->farms);
				ReduceArmForceGPU<<<blocksArms,threadsPerBlock>>>(armCount, device->g2arms_pos, device->g2arms_ind, device->f2, device->farms);
				ReduceArmForceGPU<<<blocksArms,threadsPerBlock>>>(armCount, device->g3arms_pos, device->g3arms_ind, device->f3, device->farms);
				ReduceArmForceGPU<<<blocksArms,threadsPerBlock>>>(armCount, device->g4arms_pos, device->g4arms_ind, device->f4, device->farms);
			}
			return;
		}
		
		
		/* Grab the time step */
		if (reqType == FULL || reqType == GROUP0) newDT = MIN(param->maxDT, param->nextDT    );
		else if              ( reqType == GROUP1) newDT = MIN(param->maxDT, param->nextDTsub );
		else if              ( reqType == GROUP2) newDT = MIN(param->maxDT, param->nextDTsub2);
		else if              ( reqType == GROUP3) newDT = MIN(param->maxDT, param->nextDTsub3);
		else if              ( reqType == GROUP4) newDT = MIN(param->maxDT, param->nextDTsub4);
		
		if (newDT <= 0.0) {
			if (reqType == FULL || reqType == GROUP0) newDT = param->maxDT;
			else newDT = param->realdt;
		}
		
		if (reqType == FULL || reqType == GROUP0) param->deltaTT     = newDT;
		else if              ( reqType == GROUP1) param->deltaTTsub  = newDT;
		else if              ( reqType == GROUP2) param->deltaTTsub2 = newDT;
		else if              ( reqType == GROUP3) param->deltaTTsub3 = newDT;
		else if              ( reqType == GROUP4) param->deltaTTsub4 = newDT;
		
		/* RKF integration: initialize convergence loop */
		int convergent =  0;
		int incrDelta  =  1;
		int iTry       = -1;
		
		/* If there are no interactions, just skip this loop */
		if (reqType == GROUP0 && segSegCount == 0) {
			convergent = 1;
			errMax = 0.0;
		}

		while (!convergent) {
			iTry++;

			for (i = 0; i < 5; i++) {
				
				RKFStepGPU<0,1><<<blocksNodes,threadsPerBlock>>>(i, nodeCount, newDT, device->v, device->rkf, device->r0, device->r, device->e1, device->e2, device->f, device->n);
				
				switch (param->elasticinteraction) {
					case 0:
						if (reqType == FULL || reqType == GROUP1) {
							SegForceDragGPU<0><<<blocksSegs,threadsPerBlock>>>(segCount, device->r, seg, device->b, device->esig, fseg, device->Bseg, fmm);
						}
						break;
					case 1:
						if (reqType == FULL || reqType == GROUP1) {
							SegForceDragGPU<1><<<blocksSegs,threadsPerBlock>>>(segCount, device->r, seg, device->b, device->esig, fseg, device->Bseg, fmm);
							if (reqType == FULL) {
								SegSegForceGPU<0><<<device->blocksSegSegs0,threadsPerBlock>>>(device->nSegSeg0, device->r, seg, device->b, device->g0, device->cc, gflag, device->f0);
								SegSegForceGPU<0><<<device->blocksSegSegs2,threadsPerBlock>>>(device->nSegSeg2, device->r, seg, device->b, device->g2, device->cc, gflag, device->f2);
								SegSegForceGPU<0><<<device->blocksSegSegs3,threadsPerBlock>>>(device->nSegSeg3, device->r, seg, device->b, device->g3, device->cc, gflag, device->f3);
								SegSegForceGPU<0><<<device->blocksSegSegs4,threadsPerBlock>>>(device->nSegSeg4, device->r, seg, device->b, device->g4, device->cc, gflag, device->f4);
							}
						} else {
							if (reqType == GROUP0) {
								SegSegForceGPU<1><<<blocksSegSegs,threadsPerBlock>>>(segSegCount, device->r, seg, device->b, group, device->cc, gflag, fseg);
							} else {
								SegSegForceGPU<0><<<blocksSegSegs,threadsPerBlock>>>(segSegCount, device->r, seg, device->b, group, device->cc, gflag, fseg);
							}
							SegDragGPU<<<blocksSegs,threadsPerBlock>>>(segCount, device->r, seg, device->b, device->Bseg);
						}
						break;
				}
				
				if (reqType == FULL) {
					ResetForceGPU<<<blocksNodes,threadsPerBlock>>>(nodeCount, device->f);
					ReduceForceGPU<1><<<blocksNodes,threadsPerBlock>>>(nodeCount, device->g0pos, device->g0ind, device->f0, device->f);
					ReduceForceGPU<1><<<blocksNodes,threadsPerBlock>>>(nodeCount, device->g1pos, device->g1ind, device->fseg, device->f);
					ReduceForceGPU<1><<<blocksNodes,threadsPerBlock>>>(nodeCount, device->g2pos, device->g2ind, device->f2, device->f);
					ReduceForceGPU<1><<<blocksNodes,threadsPerBlock>>>(nodeCount, device->g3pos, device->g3ind, device->f3, device->f);
					ReduceForceGPU<1><<<blocksNodes,threadsPerBlock>>>(nodeCount, device->g4pos, device->g4ind, device->f4, device->f);
				} else {
					ReduceForceGPU<0><<<blocksNodes,threadsPerBlock>>>(nodeCount, gpos, gind, fseg, device->f);
				}
				ReduceDragGPU<<<blocksNodes,threadsPerBlock>>>(nodeCount, spos, sind, device->Bseg, device->B);
				
				NodeVelocityGPU<<<blocksNodes,threadsPerBlock>>>(nodeCount, device->f, device->B, device->mob, device->n, device->v);
			}
			
			if (reqType == GROUP0 && iTry < param->nTry) {
				switch (threadsPerBlock) {
					case 1024: RKFStepGPU<1,1024><<<blocksNodes,threadsPerBlock>>>(5, nodeCount, newDT, device->v, device->rkf, device->r0, device->r, device->e1, device->e2, device->f, device->n); break;
					case 512:  RKFStepGPU<1,512><<<blocksNodes,threadsPerBlock>>>(5, nodeCount, newDT, device->v, device->rkf, device->r0, device->r, device->e1, device->e2, device->f, device->n); break;
					case 256:  RKFStepGPU<1,256><<<blocksNodes,threadsPerBlock>>>(5, nodeCount, newDT, device->v, device->rkf, device->r0, device->r, device->e1, device->e2, device->f, device->n); break;
					case 128:  RKFStepGPU<1,128><<<blocksNodes,threadsPerBlock>>>(5, nodeCount, newDT, device->v, device->rkf, device->r0, device->r, device->e1, device->e2, device->f, device->n); break;
					case 64:   RKFStepGPU<1,64><<<blocksNodes,threadsPerBlock>>>(5, nodeCount, newDT, device->v, device->rkf, device->r0, device->r, device->e1, device->e2, device->f, device->n); break;
					default:   Fatal("RKFStepGPU is not defined for threadsPerBlock = %d", threadsPerBlock); break;
				}
			} else {
				switch (threadsPerBlock) {
					case 1024: RKFStepGPU<0,1024><<<blocksNodes,threadsPerBlock>>>(5, nodeCount, newDT, device->v, device->rkf, device->r0, device->r, device->e1, device->e2, device->f, device->n); break;
					case 512:  RKFStepGPU<0,512><<<blocksNodes,threadsPerBlock>>>(5, nodeCount, newDT, device->v, device->rkf, device->r0, device->r, device->e1, device->e2, device->f, device->n); break;
					case 256:  RKFStepGPU<0,256><<<blocksNodes,threadsPerBlock>>>(5, nodeCount, newDT, device->v, device->rkf, device->r0, device->r, device->e1, device->e2, device->f, device->n); break;
					case 128:  RKFStepGPU<0,128><<<blocksNodes,threadsPerBlock>>>(5, nodeCount, newDT, device->v, device->rkf, device->r0, device->r, device->e1, device->e2, device->f, device->n); break;
					case 64:   RKFStepGPU<0,64><<<blocksNodes,threadsPerBlock>>>(5, nodeCount, newDT, device->v, device->rkf, device->r0, device->r, device->e1, device->e2, device->f, device->n); break;
					default:   Fatal("RKFStepGPU is not defined for threadsPerBlock = %d", threadsPerBlock); break;
				}
			}
			
			/* Calculate maximum error */
#if 1
			errMax = *(thrust::max_element(device->e1_ptr, device->e1_ptr + blocksNodes));
			relErrMax = *(thrust::max_element(device->e2_ptr, device->e2_ptr + blocksNodes));
#else
			double *e1 = (double*)malloc(sizeof(double)*blocksNodes); //ALLOCATE this once per time-step if used
			double *e2 = (double*)malloc(sizeof(double)*blocksNodes);
			HANDLE_ERROR(cudaMemcpy(e1, device->e1, sizeof(double)*blocksNodes, cudaMemcpyDeviceToHost));
			HANDLE_ERROR(cudaMemcpy(e2, device->e2, sizeof(double)*blocksNodes, cudaMemcpyDeviceToHost));
			errMax = 0.0;
			relErrMax = 0.0;
			for (i = 0; i < blocksNodes; i++) {
				errMax = MAX(e1[i], errMax);
				relErrMax = MAX(e2[i], relErrMax);
			}
			free(e1);
			free(e2);
#endif
			
			if (errMax < param->rTol && relErrMax < param->rTolrel) {
				convergent = 1;
				
				/* Flag oscillating nodes for subsequent cycles */
				if (reqType > GROUP0 && nSubcyc > 3) {
					ForwardProgressCheckGPU<<<blocksNodes,threadsPerBlock>>>(nodeCount, device->v0, device->v, device->n);
				}
				
			} else {
				
				if ((iTry < param->nTry) && (reqType == GROUP0)) {
					
					tmp_rg9s = rg9s * (iTry+1) * (iTry+1);
					MoveInteractionsGPU<<<blocksSegSegs,threadsPerBlock>>>(segSegCount, tmp_rg9s, seg, group, device->g0dist2, device->n, gflag);
				
				} else {
				
					// Restart with old velocities
					ResetNodesGPU<<<blocksNodes,threadsPerBlock>>>(nodeCount, device->v, device->v0);
					
					incrDelta = 0;
					newDT    *= param->dtDecrementFact;

					if ((newDT < 1.0e-20) && (home->myDomain == 0)) {
						Fatal("RKFIntegrator(): Timestep has dropped below\n"
							  "minimal threshold to %e.  Aborting!", newDT);
					}
				
				}
			}

		}
		
/*
 *      Automatically increment timestep if convergence was reached
 *      on the very first iteration of the above loop.
 *
 *      If variable timestep adjustments are enabled, calculate an
 *      adjustment factor based on the maximum allowed timestep increment
 *      and the maximum error found above.  If variable timestep
 *      adjustments are not enabled, adjust the timestep by the
 *      maximum permitted factor.
 */
		
		if (reqType == FULL || reqType == GROUP0) {
			param->deltaTT     = newDT;
			param->realdt      = newDT;
			param->timeStart   = param->timeNow;
		} else if (reqType == GROUP1) {
			param->deltaTTsub  = newDT;
			param->realdtsub   = newDT;
		} else if (reqType == GROUP2) {
			param->deltaTTsub2 = newDT;
			param->realdtsub2  = newDT;
		} else if (reqType == GROUP3) {
			param->deltaTTsub3 = newDT;
			param->realdtsub3  = newDT;
		} else if (reqType == GROUP4) {
			param->deltaTTsub4 = newDT;
			param->realdtsub4  = newDT;
		}
		
		if (incrDelta) {
			if (param->dtVariableAdjustment) {
				real8 tmp1, tmp2, tmp3, tmp4, factor;
				tmp1 = pow(param->dtIncrementFact, param->dtExponent);
				tmp2 = errMax/param->rTol;
				tmp3 = 1.0 / param->dtExponent;
				tmp4 = pow(1.0/(1.0+(tmp1-1.0)*tmp2), tmp3);
				factor = param->dtIncrementFact * tmp4;
				
				newDT = MIN(param->maxDT, newDT*factor);
			} else {
				newDT = MIN(param->maxDT, newDT*param->dtIncrementFact);
			}
		}
	
		if (reqType == FULL || reqType == GROUP0) param->nextDT     = newDT;
		else if              ( reqType == GROUP1) param->nextDTsub  = newDT;
		else if              ( reqType == GROUP2) param->nextDTsub2 = newDT;
		else if              ( reqType == GROUP3) param->nextDTsub3 = newDT;
		else if              ( reqType == GROUP4) param->nextDTsub4 = newDT;
		
}

/*------------------------------------------------------------------------
 *
 *      Function:    GetMinDistGPU
 *      Description: Calculate the minimum distance between two segments
 *
 *-----------------------------------------------------------------------*/
__device__ void GetMinDistGPU(double3 r1, double3 r2, double3 r3, double3 r4, double &dist2)
{
	int     i, pos;
	int     icase, didDist2;
	double  A, B, C, D, E;
	double  eps = 1.0e-12;
	double  distx, disty, distz, d2, d2min;
	double  dist[4], L1, L2;

	double3 seg1L, seg2L;
	double3 r1mr3, r2mr1, r4mr3, r4mr1, r3mr2, r4mr2;
	double  M[2][2], rhs[2], sol[2], detM;
	double  trial[4][2];
	
	r1mr3.x = r1.x - r3.x;
	r1mr3.y = r1.y - r3.y;
	r1mr3.z = r1.z - r3.z;
	
	r2mr1.x = r2.x - r1.x;
	r2mr1.y = r2.y - r1.y;
	r2mr1.z = r2.z - r1.z;
	
	r4mr3.x = r4.x - r3.x;
	r4mr3.y = r4.y - r3.y;
	r4mr3.z = r4.z - r3.z;
	
	seg1L = r2mr1;
	seg2L = r4mr3;
	
	M[0][0] = DotProductGPU(r2mr1, r2mr1);
	M[1][0] =-DotProductGPU(r4mr3, r2mr1);
	M[1][1] = DotProductGPU(r4mr3, r4mr3);
	M[0][1] = M[1][0];
	
	rhs[0] = -DotProductGPU(r2mr1, r1mr3);
	rhs[1] =  DotProductGPU(r4mr3, r1mr3);
	
	detM = 1.0 - M[1][0] * M[1][0] / M[0][0] / M[1][1];

	A = M[0][0];
	B = -2.0 * rhs[0];
	C = -2.0 * M[1][0];
	D = -2.0 * rhs[1];
	E = M[1][1];

	didDist2 = 0;
	
/*
 *      If segment 1 is just a point...
 */
        if (A < eps) {
            L1 = 0.0;
            if (E < eps) L2 = 0.0;
            else L2 = -0.5 * D / E;

/*
 *      If segment 2 is just a point...
 */
        } else if (E < eps) {
            L2 = 0.0;
            if (A < eps) L1 = 0.0;
            else L1 = -0.5 * B / A;
/*
 *      If segments are parallel
 */
		} else if (detM<1e-6) {
			
			r4mr1.x = r4.x - r1.x;
			r4mr1.y = r4.y - r1.y;
			r4mr1.z = r4.z - r1.z;
			
			r3mr2.x = r3.x - r2.x;
			r3mr2.y = r3.y - r2.y;
			r3mr2.z = r3.z - r2.z;
			
			r4mr2.x = r4.x - r2.x;
			r4mr2.y = r4.y - r2.y;
			r4mr2.z = r4.z - r2.z;
			
			dist[0] = DotProductGPU(r1mr3, r1mr3);
            dist[1] = DotProductGPU(r4mr1, r4mr1);
            dist[2] = DotProductGPU(r3mr2, r3mr2);
            dist[3] = DotProductGPU(r4mr2, r4mr2);

            dist2 = dist[0];
            pos = 1;

            for (i = 1; i < 4; i++) {
                if (dist[i] < dist2) {
                    dist2 = dist[i];
                    pos = i+1;
                }
            }

            L1 = floor((double)pos/2.1);
			L2 = (double)(1 - (pos % 2));
			didDist2 = 1;
/*
 *		Solve the general case
 */
		} else { 
			detM *= M[0][0]*M[1][1];
			sol[0] = ( M[1][1]*rhs[0] - M[0][1]*rhs[1]) / detM;
			sol[1] = (-M[1][0]*rhs[0] + M[0][0]*rhs[1]) / detM;

			if ((sol[0]>=0) && (sol[0]<=1) && (sol[1]>=0) && (sol[1]<=1)) {
				/* we are done here */
				L1 = sol[0];
				L2 = sol[1];

			} else {

				/* enumerate four cases */
				/* alpha = 0 */
				icase = 0;
				trial[icase][0] = 0;
				trial[icase][1] = (rhs[1] - M[1][0]*trial[icase][0]) / M[1][1];

				/* alpha = 1 */
				icase = 1;
				trial[icase][0] = 1;
				trial[icase][1] = (rhs[1] - M[1][0]*trial[icase][0]) / M[1][1];

				/* beta = 0 */
				icase = 2;
				trial[icase][1] = 0;
				trial[icase][0] = (rhs[0] - M[0][1]*trial[icase][1]) / M[0][0];

				/* beta = 1 */
				icase = 3;
				trial[icase][1] = 1;
				trial[icase][0] = (rhs[0] - M[0][1]*trial[icase][1]) / M[0][0];

				/* find the minimum out of four trials */
				d2min = 1e100;
				for(icase = 0; icase < 4; icase++) {
					trial[icase][0] = min(max(trial[icase][0], 0.0), 1.0);
					trial[icase][1] = min(max(trial[icase][1], 0.0), 1.0);
					distx = r1.x + (seg1L.x * trial[icase][0]) 
					      - r3.x - (seg2L.x * trial[icase][1]);
					disty = r1.y + (seg1L.y * trial[icase][0]) 
					      - r3.y - (seg2L.y * trial[icase][1]);
					distz = r1.z + (seg1L.z * trial[icase][0]) 
					      - r3.z - (seg2L.z * trial[icase][1]);

					d2 = distx*distx + disty*disty + distz*distz;
					if (d2<d2min) {
						L1 = trial[icase][0];
						L2 = trial[icase][1];
						d2min = d2;
					}
				}
				dist2 = d2min;
				didDist2 = 1;
			}
		} 

/*
 *      Make sure L1 and L2 are between 0 and 1
 */
        L1 = min(max(L1, 0.0), 1.0);
        L2 = min(max(L2, 0.0), 1.0);

		if (!didDist2) {
			distx = r1.x + (seg1L.x * L1) - r3.x - (seg2L.x * L2);
			disty = r1.y + (seg1L.y * L1) - r3.y - (seg2L.y * L2);
			distz = r1.z + (seg1L.z * L1) - r3.z - (seg2L.z * L2);

			dist2 = distx*distx + disty*disty + distz*distz;
		}
	
	return;
}

/*------------------------------------------------------------------------
 *
 *      Function:    GetMinDistSegSegGPU
 *      Description: Determine segment pairs interaction group based
 *                   on the segment / segment distance
 *
 *-----------------------------------------------------------------------*/
__global__ void GetMinDistSegSegGPU(int segSegCount, double3 *r, int2 *s, int2 *g, int *rg, double *gdist2, int *gflag)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < segSegCount) {
		
		double dist2;
		
		int2 segs = g[i];
		
		int2 s1 = s[segs.x];
		int2 s2 = s[segs.y];
		
		double3 r1, r2, r3, r4;
		
		int n1 = s1.x;
		int n2 = s1.y;
		int n3 = s2.x;
		int n4 = s2.y;
		
		r1 = r[n1];
		r2 = r[n2];
		r3 = r[n3];
		r4 = r[n4];
		
		double3 dr1, dr2;
		double  L1s, L2s;
		dr1.x = r2.x - r1.x;
		dr1.y = r2.y - r1.y;
		dr1.z = r2.z - r1.z;
		ZImageGPU(dr1);
		L1s = dr1.x*dr1.x + dr1.y*dr1.y + dr1.z*dr1.z;
		
		dr2.x = r4.x - r3.x;
		dr2.y = r4.y - r3.y;
		dr2.z = r4.z - r3.z;
		ZImageGPU(dr2);
		L2s = dr2.x*dr2.x + dr2.y*dr2.y + dr2.z*dr2.z;
		
		if (L1s < 1.e-20 || L2s < 1.e-20) {
			dist2 = -1.0;
		} else {
			
			r2.x = r1.x + dr1.x;
			r2.y = r1.y + dr1.y;
			r2.z = r1.z + dr1.z;
			
			dr1.x = r3.x - r1.x;
			dr1.y = r3.y - r1.y;
			dr1.z = r3.z - r1.z;
			ZImageGPU(dr1);
			
			r3.x = r1.x + dr1.x;
			r3.y = r1.y + dr1.y;
			r3.z = r1.z + dr1.z;
			
			dr2.x = r4.x - r3.x;
			dr2.y = r4.y - r3.y;
			dr2.z = r4.z - r3.z;
			ZImageGPU(dr2);
			
			r4.x = r3.x + dr2.x;
			r4.y = r3.y + dr2.y;
			r4.z = r3.z + dr2.z;
			
			int hinge = 0;
			if (n1 == n3) {
				hinge = 1;
			} else if (n2 == n3) {
				hinge = 2;
			} else if (n2 == n4) {
				hinge = 3;
			} else if (n1 == n4) {
				hinge = 4;
			}
			
			if (!hinge) {
				GetMinDistGPU(r1, r2, r3, r4, dist2);
			} else {
				
				double3 m, h1, h2;
				
				if (hinge == 1) {
					m = r1;
					h1 = r2;
					h2 = r4;
					
				} else if (hinge == 2) {
					m = r2;
					h1 = r1;
					h2 = r4;
					
				} else if (hinge == 3) {
					m = r2;
					h1 = r1;
					h2 = r3;
					
				} else if (hinge == 4) {
					m = r1;
					h1 = r2;
					h2 = r3;
				}
				
				if (L1s>L2s) {
					GetMinDistGPU(m , h1, h2, h2, dist2); 
				} else {
					GetMinDistGPU(m, h2, h1, h1, dist2); 
				}
			}
			
		}
		
		/*
		if ((*dist2 > param->cutoff2 * param->cutoff2) &&
		    (param->forceCutOff)) *dist2 = -1.0;
		*/
		
		int SubGroup;
		if (dist2 < 0) {
			SubGroup = -1;
		} else if (dist2 < rg[0]) {
			SubGroup = 1;
		} else if (dist2 < rg[1]) {
			SubGroup = 2;
		} else if (dist2 < rg[2]) {
			SubGroup = 3;
		} else if (dist2 < rg[3]) {
			SubGroup = 4;
		} else {
			SubGroup = 0;
		}
		
		gflag[i] = SubGroup;
		gdist2[i] = dist2;
	}
}

/*------------------------------------------------------------------------
 *
 *      Function:    SegSegListGPU
 *      Description: Assign each segment pair interaction to a subcycle
 *                   group based on their interaction distance
 *
 *-----------------------------------------------------------------------*/
void SegSegListGPU(Home_t *home, Device_t *device)
{
		int segSegCount, blocksSegSegs, threadsPerBlock;
		Param_t *param = home->param;
		
		segSegCount     = device->nSegSeg0;
		blocksSegSegs   = device->blocksSegSegs0;
		threadsPerBlock = device->threadsPerBlock;
		
		int2 *seg = device->g1;
		
		int rg_host[4], *rg_device;
		rg_host[0] = param->rg1 * param->rg1;
		rg_host[1] = param->rg2 * param->rg2;
		rg_host[2] = param->rg3 * param->rg3;
		rg_host[3] = param->rg4 * param->rg4;
		
		HANDLE_ERROR(cudaMalloc(&rg_device, sizeof(int)*4));
		HANDLE_ERROR(cudaMemcpy(rg_device, rg_host, sizeof(int)*4, cudaMemcpyHostToDevice));
		
		GetMinDistSegSegGPU<<<blocksSegSegs,threadsPerBlock>>>(segSegCount, device->r, seg, device->g0, rg_device, device->g0dist2, device->g0flag);
		
		HANDLE_ERROR(cudaFree(rg_device));
		
		return;
}

/*------------------------------------------------------------------------
 *
 *      Function:    PackReductionGPU
 * 		Description: Pack the reduction array used to assemble nodal
 * 					 forces from GPU forces array.
 *
 *-----------------------------------------------------------------------*/
__global__ void PackReductionGPU(int segSegCount, int2 *g, int2 *s, int *nind, int *aind, int2 *gpos, int2 *garms_pos, int *gind, int *garms_ind)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < segSegCount) {
		
		int i1, i2, i3, i4;
		int pos1, pos2, pos3, pos4;
		
		int2 segs = g[i];
		int2 s1 = s[segs.x];
		int2 s2 = s[segs.y];
		
		i1 = s1.x;
		i2 = s1.y;
		i3 = s2.x;
		i4 = s2.y;
		
		pos1 = gpos[i1].x + nind[4*i+0];
		pos2 = gpos[i2].x + nind[4*i+1];
		pos3 = gpos[i3].x + nind[4*i+2];
		pos4 = gpos[i4].x + nind[4*i+3];
		
		gind[pos1] = i*4+0;
		gind[pos2] = i*4+1;
		gind[pos3] = i*4+2;
		gind[pos4] = i*4+3;
		
		i1 = 2*segs.x+0;
		i2 = 2*segs.x+1;
		i3 = 2*segs.y+0;
		i4 = 2*segs.y+1;
		
		pos1 = garms_pos[i1].x + aind[4*i+0];
		pos2 = garms_pos[i2].x + aind[4*i+1];
		pos3 = garms_pos[i3].x + aind[4*i+2];
		pos4 = garms_pos[i4].x + aind[4*i+3];
		
		garms_ind[pos1] = i*4+0;
		garms_ind[pos2] = i*4+1;
		garms_ind[pos3] = i*4+2;
		garms_ind[pos4] = i*4+3;
		
	}
}

/*------------------------------------------------------------------------
 *
 *      Function:    SendSubgroupGPU
 *      Description: Pack, allocate and send segments / segments 
 *                   information for the different groups on the GPU.
 *
 *-----------------------------------------------------------------------*/
void SendSubgroupGPU(Home_t *home, Device_t *device, int subGroup)
{
		int         i, j, k, nodeCount, armCount, nSegSeg;
		int         s1, s2, n1, n2, n3, n4, a1, a2, a3, a4;
		Node_t      *node1, *node2, *node3, *node4;
		SegSeg_t    *SegSegList;
		Subcyc_t    *subcyc;
		
		subcyc  = home->subcyc;
		nodeCount = device->nodeCount;
		armCount = device->armCount;
		
/*
 *		Group1 (segments) packing for GPU has already be done at this stage
 */		
		if (subGroup == GROUP1) {
			return;
		}
		
/*
 *		Pack groups and segment / segment interactions for GPU
 */		
		//if (param->elasticinteraction) {
				
			if (subGroup == GROUP0) {
				SegSegList = subcyc->SegSegListG0;
				nSegSeg    = subcyc->SegSegListG0_cnt;
			} else if (subGroup == GROUP2) {
				SegSegList = subcyc->SegSegListG2;
				nSegSeg    = subcyc->SegSegListG2_cnt;
			} else if (subGroup == GROUP3) {
				SegSegList = subcyc->SegSegListG3;
				nSegSeg    = subcyc->SegSegListG3_cnt;
			} else if (subGroup == GROUP4) {
				SegSegList = subcyc->SegSegListG4;
				nSegSeg    = subcyc->SegSegListG4_cnt;
			}
			
			int cntSegSeg = 0;
			for (j = 0; j < nSegSeg; j++) {
				if (SegSegList[j].flag == 0) continue;
				cntSegSeg++;
			}
			
			// Check block size
			int blocksSegSegs = (cntSegSeg + device->threadsPerBlock - 1) / device->threadsPerBlock;
			if (blocksSegSegs > home->deviceProp->maxBlocks) {
				Fatal("Max number of CUDA blocks exceeded for group %d!", subGroup-GROUP0);
			}
			
			
			int2 *gList = (int2*)malloc(cntSegSeg*sizeof(int2));
			
			int2 *redg_pos = (int2*)malloc(nodeCount*sizeof(int2));
			int  *redg_ind = (int*)malloc(4*cntSegSeg*sizeof(int));
			
			int2 *garm_pos = (int2*)malloc(armCount*sizeof(int2));
			int  *garm_ind = (int*)malloc(4*cntSegSeg*sizeof(int));
			
			
			if (subGroup == GROUP0) {			
/*				
 * 				Initially, all segments / segments interactions are in group 0
 * 				when using the GPU subcycle integrator (this is to maximize
 * 				code performance). Pack the interactions information and store
 * 				the positions of the interactions forces to be retreived from the
 * 				GPU force array for each node in the reduction array. In the case 
 * 				of group 0, the reduction array is built on the GPU to gain time.
 */
				int npos = 0;
				int apos = 0;
				int nCount = 0;
				
				for (i = 0; i < home->newNodeKeyPtr; i++) {
					if ((node1 = home->nodeKeys[i]) == (Node_t *)NULL) continue;
					redg_pos[nCount].x = npos;
					npos += node1->numInt;
					redg_pos[nCount].y = npos;
					nCount++;
				}
				for (j = 0; j < subcyc->SegListG1_cnt; j++) {
					node1 = subcyc->SegListG1[j].seg->node1;
					node2 = subcyc->SegListG1[j].seg->node2;
					a1 = subcyc->SegListG1[j].seg->armID12;
					a2 = subcyc->SegListG1[j].seg->armID21;
					
					garm_pos[2*j+0].x = apos;
					apos += node1->armInt[a1];
					garm_pos[2*j+0].y = apos;
					node1->armInt[a1] = 0;
					
					garm_pos[2*j+1].x = apos;
					apos += node2->armInt[a2];
					garm_pos[2*j+1].y = apos;
					node2->armInt[a2] = 0;
				}
				
				cntSegSeg = 0;
				for (j = 0; j < nSegSeg; j++) {
					if (SegSegList[j].flag == 0) continue;
					gList[cntSegSeg].x = SegSegList[j].seg1->subindex;
					gList[cntSegSeg].y = SegSegList[j].seg2->subindex;
					cntSegSeg++;
				}
				
				// Allocate memory on device
				device->nSegSeg0 = cntSegSeg;
				device->blocksSegSegs0 = blocksSegSegs;
				
				HANDLE_ERROR(cudaMalloc(&device->f0, sizeof(double3)*4*cntSegSeg));
				HANDLE_ERROR(cudaMalloc(&device->g0, sizeof(int2)*cntSegSeg));
				HANDLE_ERROR(cudaMalloc(&device->g0pos, sizeof(int2)*nodeCount));
				HANDLE_ERROR(cudaMalloc(&device->g0ind, sizeof(int)*4*cntSegSeg));
				HANDLE_ERROR(cudaMalloc(&device->g0arms_pos, sizeof(int2)*armCount));
				HANDLE_ERROR(cudaMalloc(&device->g0arms_ind, sizeof(int)*4*cntSegSeg));
				HANDLE_ERROR(cudaMalloc(&device->g0flag, sizeof(int)*cntSegSeg));
				HANDLE_ERROR(cudaMalloc(&device->g0dist2, sizeof(double)*cntSegSeg));
				
				int *device_nind, *device_aind;
				HANDLE_ERROR(cudaMalloc(&device_nind, sizeof(int)*4*cntSegSeg));
				HANDLE_ERROR(cudaMalloc(&device_aind, sizeof(int)*4*cntSegSeg));
				
				HANDLE_ERROR(cudaMemcpy(device->g0, gList, sizeof(int2)*cntSegSeg, cudaMemcpyHostToDevice));
				HANDLE_ERROR(cudaMemcpy(device->g0pos, redg_pos, sizeof(int2)*nodeCount, cudaMemcpyHostToDevice));
				HANDLE_ERROR(cudaMemcpy(device->g0arms_pos, garm_pos, sizeof(int2)*armCount, cudaMemcpyHostToDevice));
				HANDLE_ERROR(cudaMemcpy(device_nind, subcyc->SegSegListNodeInd, sizeof(int)*4*cntSegSeg, cudaMemcpyHostToDevice));
				HANDLE_ERROR(cudaMemcpy(device_aind, subcyc->SegSegListArmInd, sizeof(int)*4*cntSegSeg, cudaMemcpyHostToDevice));
				
				// Reduction kernel
				PackReductionGPU<<<blocksSegSegs,device->threadsPerBlock>>>(cntSegSeg, device->g0, device->g1, device_nind, device_aind, device->g0pos, device->g0arms_pos, device->g0ind, device->g0arms_ind);
				
				HANDLE_ERROR(cudaFree(device_nind));
				HANDLE_ERROR(cudaFree(device_aind));
				
			} else {
/*			
 * 				For other groups, pack both the segment pairs and 
 * 				the reduction array on the CPU.
 */
				std::vector<std::vector<int> > redg(nodeCount);
				std::vector<std::vector<int> > redgarms(armCount);

				cntSegSeg = 0;
				for (j = 0; j < nSegSeg; j++) {
					if (SegSegList[j].flag == 0) continue;
					
					s1 = SegSegList[j].seg1->subindex;
					s2 = SegSegList[j].seg2->subindex;
					
					gList[cntSegSeg].x = s1;
					gList[cntSegSeg].y = s2;
					
					node1 = SegSegList[j].seg1->node1;
					node2 = SegSegList[j].seg1->node2;
					node3 = SegSegList[j].seg2->node1;
					node4 = SegSegList[j].seg2->node2;
					
					n1 = node1->subindex;
					n2 = node2->subindex;
					n3 = node3->subindex;
					n4 = node4->subindex;
					
					redg[n1].push_back(cntSegSeg*4+0);
					redg[n2].push_back(cntSegSeg*4+1);
					redg[n3].push_back(cntSegSeg*4+2);
					redg[n4].push_back(cntSegSeg*4+3);
					
					a1 = SegSegList[j].seg1->armID12;
					a2 = SegSegList[j].seg1->armID21;
					a3 = SegSegList[j].seg2->armID12;
					a4 = SegSegList[j].seg2->armID21;
					
					redgarms[node1->armid[a1]].push_back(cntSegSeg*4+0);
					redgarms[node2->armid[a2]].push_back(cntSegSeg*4+1);
					redgarms[node3->armid[a3]].push_back(cntSegSeg*4+2);
					redgarms[node4->armid[a4]].push_back(cntSegSeg*4+3);
					
					cntSegSeg++;
				}
			
				// Pack reduction array
				int ind = 0;
				for (j = 0; j < nodeCount; j++) {
					redg_pos[j].x = ind;
					for (k = 0; k < redg[j].size(); k++) {
						redg_ind[ind++] = redg[j][k];
					}
					redg_pos[j].y = ind;
				}
				if (ind != 4*cntSegSeg) Fatal("Group%d reduction array size error!", subGroup-GROUP0);
			
				// Pack arm reduction array
				ind = 0;
				for (j = 0; j < armCount; j++) {
					garm_pos[j].x = ind;
					for (k = 0; k < redgarms[j].size(); k++) {
						garm_ind[ind++] = redgarms[j][k];
					}
					garm_pos[j].y = ind;
				}
				if (ind != 4*cntSegSeg) Fatal("Group%d arm reduction array size error!", subGroup-GROUP0);
			
			}
			
			// Allocate memory on device
			if (subGroup == GROUP2) {
				device->nSegSeg2 = cntSegSeg;
				device->blocksSegSegs2 = blocksSegSegs;
			
				HANDLE_ERROR(cudaMalloc(&device->f2, sizeof(double3)*4*cntSegSeg));
				HANDLE_ERROR(cudaMalloc(&device->g2, sizeof(int2)*cntSegSeg));
				HANDLE_ERROR(cudaMalloc(&device->g2pos, sizeof(int2)*nodeCount));
				HANDLE_ERROR(cudaMalloc(&device->g2ind, sizeof(int)*4*cntSegSeg));
				HANDLE_ERROR(cudaMalloc(&device->g2arms_pos, sizeof(int2)*armCount));
				HANDLE_ERROR(cudaMalloc(&device->g2arms_ind, sizeof(int)*4*cntSegSeg));
				
				HANDLE_ERROR(cudaMemcpy(device->g2, gList, sizeof(int2)*cntSegSeg, cudaMemcpyHostToDevice));
				HANDLE_ERROR(cudaMemcpy(device->g2pos, redg_pos, sizeof(int2)*nodeCount, cudaMemcpyHostToDevice));
				HANDLE_ERROR(cudaMemcpy(device->g2ind, redg_ind, sizeof(int)*4*cntSegSeg, cudaMemcpyHostToDevice));
				HANDLE_ERROR(cudaMemcpy(device->g2arms_pos, garm_pos, sizeof(int2)*armCount, cudaMemcpyHostToDevice));
				HANDLE_ERROR(cudaMemcpy(device->g2arms_ind, garm_ind, sizeof(int)*4*cntSegSeg, cudaMemcpyHostToDevice));
			
			} else if (subGroup == GROUP3) {
				device->nSegSeg3 = cntSegSeg;
				device->blocksSegSegs3 = blocksSegSegs;
			
				HANDLE_ERROR(cudaMalloc(&device->f3, sizeof(double3)*4*cntSegSeg));
				HANDLE_ERROR(cudaMalloc(&device->g3, sizeof(int2)*cntSegSeg));
				HANDLE_ERROR(cudaMalloc(&device->g3pos, sizeof(int2)*nodeCount));
				HANDLE_ERROR(cudaMalloc(&device->g3ind, sizeof(int)*4*cntSegSeg));
				HANDLE_ERROR(cudaMalloc(&device->g3arms_pos, sizeof(int2)*armCount));
				HANDLE_ERROR(cudaMalloc(&device->g3arms_ind, sizeof(int)*4*cntSegSeg));
				
				HANDLE_ERROR(cudaMemcpy(device->g3, gList, sizeof(int2)*cntSegSeg, cudaMemcpyHostToDevice));
				HANDLE_ERROR(cudaMemcpy(device->g3pos, redg_pos, sizeof(int2)*nodeCount, cudaMemcpyHostToDevice));
				HANDLE_ERROR(cudaMemcpy(device->g3ind, redg_ind, sizeof(int)*4*cntSegSeg, cudaMemcpyHostToDevice));
				HANDLE_ERROR(cudaMemcpy(device->g3arms_pos, garm_pos, sizeof(int2)*armCount, cudaMemcpyHostToDevice));
				HANDLE_ERROR(cudaMemcpy(device->g3arms_ind, garm_ind, sizeof(int)*4*cntSegSeg, cudaMemcpyHostToDevice));
			
			} else if (subGroup == GROUP4) {
				device->nSegSeg4 = cntSegSeg;
				device->blocksSegSegs4 = blocksSegSegs;
				
				HANDLE_ERROR(cudaMalloc(&device->f4, sizeof(double3)*4*cntSegSeg));
				HANDLE_ERROR(cudaMalloc(&device->g4, sizeof(int2)*cntSegSeg));
				HANDLE_ERROR(cudaMalloc(&device->g4pos, sizeof(int2)*nodeCount));
				HANDLE_ERROR(cudaMalloc(&device->g4ind, sizeof(int)*4*cntSegSeg));
				HANDLE_ERROR(cudaMalloc(&device->g4arms_pos, sizeof(int2)*armCount));
				HANDLE_ERROR(cudaMalloc(&device->g4arms_ind, sizeof(int)*4*cntSegSeg));
				
				HANDLE_ERROR(cudaMemcpy(device->g4, gList, sizeof(int2)*cntSegSeg, cudaMemcpyHostToDevice));
				HANDLE_ERROR(cudaMemcpy(device->g4pos, redg_pos, sizeof(int2)*nodeCount, cudaMemcpyHostToDevice));
				HANDLE_ERROR(cudaMemcpy(device->g4ind, redg_ind, sizeof(int)*4*cntSegSeg, cudaMemcpyHostToDevice));
				HANDLE_ERROR(cudaMemcpy(device->g4arms_pos, garm_pos, sizeof(int2)*armCount, cudaMemcpyHostToDevice));
				HANDLE_ERROR(cudaMemcpy(device->g4arms_ind, garm_ind, sizeof(int)*4*cntSegSeg, cudaMemcpyHostToDevice));
			}
			
			free(gList);
			free(redg_pos);
			free(redg_ind);
			free(garm_pos);
			free(garm_ind);
		//}
		
		return;
}

/*------------------------------------------------------------------------
 *
 *      Function:    MoveInteractionGroup
 *      Description: Move a given segment pair interaction from group 0
 *                   to another group
 *
 *-----------------------------------------------------------------------*/
void MoveInteractionGroup(Subcyc_t *subcyc, int SegSegID, int subGroup)
{
		
		if (subGroup == 1) {
			Fatal("Should not move interaction to group %d!", subGroup);
			
		} else if (subGroup == 2) {
			
			if (subcyc->SegSegListG2 == NULL || 
				subcyc->SegSegListG2_cnt >= subcyc->SegSegListG2_siz) {
				subcyc->SegSegListG2_siz += 1000;
				subcyc->SegSegListG2 = (SegSeg_t *)realloc(subcyc->SegSegListG2,
										sizeof(SegSeg_t) * subcyc->SegSegListG2_siz);
			}
			
			subcyc->SegSegListG2[subcyc->SegSegListG2_cnt].seg1  = subcyc->SegSegListG0[SegSegID].seg1;
			subcyc->SegSegListG2[subcyc->SegSegListG2_cnt].seg2  = subcyc->SegSegListG0[SegSegID].seg2;
			subcyc->SegSegListG2[subcyc->SegSegListG2_cnt].flag  = 1;
			subcyc->SegSegListG2[subcyc->SegSegListG2_cnt].setSeg1Forces = 1;
			subcyc->SegSegListG2[subcyc->SegSegListG2_cnt].setSeg2Forces = 1;
			subcyc->SegSegListG2_cnt++;
			
		} else if (subGroup == 3) {
			
			if (subcyc->SegSegListG3 == NULL || 
				subcyc->SegSegListG3_cnt >= subcyc->SegSegListG3_siz) {
				subcyc->SegSegListG3_siz += 1000;
				subcyc->SegSegListG3 = (SegSeg_t *)realloc(subcyc->SegSegListG3,
										sizeof(SegSeg_t) * subcyc->SegSegListG3_siz);
			}
			
			subcyc->SegSegListG3[subcyc->SegSegListG3_cnt].seg1  = subcyc->SegSegListG0[SegSegID].seg1;
			subcyc->SegSegListG3[subcyc->SegSegListG3_cnt].seg2  = subcyc->SegSegListG0[SegSegID].seg2;
			subcyc->SegSegListG3[subcyc->SegSegListG3_cnt].flag  = 1;
			subcyc->SegSegListG3[subcyc->SegSegListG3_cnt].setSeg1Forces = 1;
			subcyc->SegSegListG3[subcyc->SegSegListG3_cnt].setSeg2Forces = 1;
			subcyc->SegSegListG3_cnt++;
			
		} else if (subGroup == 4) {
			
			if (subcyc->SegSegListG4 == NULL || 
				subcyc->SegSegListG4_cnt >= subcyc->SegSegListG4_siz) {
				subcyc->SegSegListG4_siz += 1000;
				subcyc->SegSegListG4 = (SegSeg_t *)realloc(subcyc->SegSegListG4,
										sizeof(SegSeg_t) * subcyc->SegSegListG4_siz);
			}
			
			subcyc->SegSegListG4[subcyc->SegSegListG4_cnt].seg1  = subcyc->SegSegListG0[SegSegID].seg1;
			subcyc->SegSegListG4[subcyc->SegSegListG4_cnt].seg2  = subcyc->SegSegListG0[SegSegID].seg2;
			subcyc->SegSegListG4[subcyc->SegSegListG4_cnt].flag  = 1;
			subcyc->SegSegListG4[subcyc->SegSegListG4_cnt].setSeg1Forces = 1;
			subcyc->SegSegListG4[subcyc->SegSegListG4_cnt].setSeg2Forces = 1;
			subcyc->SegSegListG4_cnt++;
			
		} else {
			Fatal("Unknown subGroup %d!", subGroup);
		}
							
		return;
}

/*------------------------------------------------------------------------
 *
 *      Function:    SubcycleIntegratorGPU
 *      Description: Perform sublcycling time-integration on the GPU
 *
 *-----------------------------------------------------------------------*/
void SubcycleIntegratorGPU(Home_t *home)
{
		int       i, j, k, l, n1, n2, mobErr;
		double    mobMatrix[3][3];
		Node_t    *node1, *node2;
		Param_t   *param;
		Subcyc_t  *subcyc;
		
		param = home->param;
		subcyc  = home->subcyc;
		
		
		Device_t *device;
		device = (Device_t*)malloc(sizeof(Device_t));
		
		cudaEvent_t start, stop;
		HANDLE_ERROR(cudaEventCreate(&start));
		HANDLE_ERROR(cudaEventCreate(&stop));
		HANDLE_ERROR(cudaEventRecord(start, 0));
		
/*
 *		Set external stress to GPU
 */		
		HANDLE_ERROR(cudaMalloc(&device->esig, sizeof(double)*6));
		HANDLE_ERROR(cudaMemcpy(device->esig, param->appliedStress, sizeof(double)*6, cudaMemcpyHostToDevice));
		
/*
 *		Pack nodes and segments for GPU
 */		
		int nodeCount = 0;
		int segCount = 0;
		int armCount = 0;
		
		for (i = 0; i < home->newNodeKeyPtr; i++) {
			if ((node1 = home->nodeKeys[i]) == (Node_t *)NULL) continue;
			armCount += node1->numNbrs;
			for (j = 0; j < node1->numNbrs; j++) {
				node2 = GetNeighborNode(home, node1, j);
				if (node2 == (Node_t *)NULL) continue;
				if (OrderNodes(node2, node1) != 1) continue;
				segCount++;
			}
			/* id in the GPU node vector */
			node1->subindex = nodeCount;
			nodeCount++;
		}
		if (armCount != 2*segCount) Fatal("armCount != 2*segCount");
		
		
/*
 * 		Determine the number of blocks and threads on which the
 * 		GPU kernels will be executed. Make sure the number of blocks 
 * 		does not exceed the hard limit on GPU devices with compute 
 * 		capability <= 3.x (maxBlocks = 65535).
 */
		int threadsPerBlock, threadsSize[8], maxThreadsSize, blockSize;
		
		threadsSize[0] = nodeCount;
		threadsSize[1] = segCount;
		threadsSize[2] = armCount;
		threadsSize[3] = subcyc->SegListG1_cnt;
		threadsSize[4] = subcyc->SegSegListG0_cnt;
		threadsSize[5] = subcyc->SegSegListG2_cnt;
		threadsSize[6] = subcyc->SegSegListG3_cnt;
		threadsSize[7] = subcyc->SegSegListG4_cnt;
		maxThreadsSize = *std::max_element(threadsSize, threadsSize+8);
		
		GetThreadsPerBlock(home->deviceProp, maxThreadsSize, &threadsPerBlock, &blockSize);
		
		if (blockSize > home->deviceProp->maxBlocks) {
			Fatal("Max number of CUDA blocks exceeded!");
		} else {
			//printf("GPU: threadsPerBlock = %d, blockSize = %d\n", threadsPerBlock, blockSize);
		}
		
		device->threadsPerBlock = threadsPerBlock;
		int blocksNodes = (nodeCount + threadsPerBlock - 1) / threadsPerBlock;
		int blocksSegs = (segCount + threadsPerBlock - 1) / threadsPerBlock;
		int blocksArms = (armCount + threadsPerBlock - 1) / threadsPerBlock;
		
		
		device->nodeCount = nodeCount;
		device->segCount = segCount;
		device->armCount = armCount;
		device->blocksNodes = blocksNodes;
		device->blocksSegs = blocksSegs;
		device->blocksArms = blocksArms;
		
		double3 *b = (double3*)malloc(segCount*sizeof(double3));
		double3 *r = (double3*)malloc(nodeCount*sizeof(double3));
		double  *mob = (double*)malloc(9*nodeCount*sizeof(double));
		int     *armid = (int*)malloc(sizeof(int)*4*segCount);
		
		double3 *cc = (double3*)malloc(nodeCount*sizeof(double3));
		int     cellX, cellY, cellZ;
        double  xCenter, yCenter, zCenter;
        Cell_t  *cell;
        
		
		nodeCount = 0;
		for (i = 0; i < home->newNodeKeyPtr; i++) {
			if ((node1 = home->nodeKeys[i]) == (Node_t *)NULL) continue;
			
			r[nodeCount].x = node1->x;
			r[nodeCount].y = node1->y;
			r[nodeCount].z = node1->z;
			
			/* Mobility matrix */
			mobErr = Mobility_FCC_0_matrix_GPU(home, node1, mobMatrix);
			for (k = 0; k < 3; k++)
				for (l = 0; l < 3; l++)
					mob[nodeCount*9+k*3+l] = mobMatrix[k][l];
			
			/* Find cell center */ // WARNING
			cell = home->cellKeys[node1->cellIdx];
			cellX = cell->xIndex;
			cellY = cell->yIndex;
			cellZ = cell->zIndex;
			FindCellCenter(param, (real8)(cellX-1), (real8)(cellY-1),
                          (real8)(cellZ-1), 2, &xCenter, &yCenter, &zCenter);
			cc[nodeCount].x = xCenter;
			cc[nodeCount].y = yCenter;
			cc[nodeCount].z = zCenter;
			/* Find cell center */
			
			nodeCount++;
		}
		
		HANDLE_ERROR(cudaMalloc(&device->cc, sizeof(double3)*nodeCount));
		HANDLE_ERROR(cudaMemcpy(device->cc, cc, sizeof(double3)*nodeCount, cudaMemcpyHostToDevice));
		
/*
 *		Pack remote force information (FMM) for GPU
 */	
		if (param->elasticinteraction && param->forceCutOff == 0 && param->fmEnabled) {
			
			subcyc->sigbFMM = (real8**)malloc(segCount*sizeof(real8*));
			RemoteSigbSub(home);
			
			int numPoints = param->fmNumPoints;
			if (numPoints != 3) Fatal("fmNumPoints needs to be 3 for GPU subcycling");
			
			double3 *fmm = (double3*)malloc(segCount*numPoints*sizeof(double3));
			
			for (i = 0; i < segCount; i++) {
				for (j = 0; j < numPoints; j++) {
					fmm[i*numPoints+j].x = subcyc->sigbFMM[i][j*3+0];
					fmm[i*numPoints+j].y = subcyc->sigbFMM[i][j*3+1];
					fmm[i*numPoints+j].z = subcyc->sigbFMM[i][j*3+2];
				}
			}
			
			HANDLE_ERROR(cudaMalloc(&device->fmm, sizeof(double3)*segCount*numPoints));
			HANDLE_ERROR(cudaMemcpy(device->fmm, fmm, sizeof(double3)*segCount*numPoints, cudaMemcpyHostToDevice));
			
			free(fmm);
			for (i = 0; i < subcyc->SegListG1_cnt; i++) {
				free(subcyc->sigbFMM[i]);
			}
			free(subcyc->sigbFMM);
		}
		
/*
 *		Pack group1 and segments for GPU
 */		
		if (segCount != subcyc->SegListG1_cnt) Fatal("segCount != subcyc->SegListG1_cnt");
		Segm_t *SegList = subcyc->SegListG1;
		
		std::vector<std::vector<int> > redg1(nodeCount);
		int2 *g1List = (int2*)malloc(segCount*sizeof(int2));
		int  *g1arms = (int*)malloc(armCount*sizeof(int));
		
		for (j = 0; j < subcyc->SegListG1_cnt; j++) {
			
			node1 = SegList[j].seg->node1;
			node2 = SegList[j].seg->node2;
			
			n1 = node1->subindex;
			n2 = node2->subindex;
			
			redg1[n1].push_back(j*2);
			redg1[n2].push_back(j*2+1);
			
			g1List[j].x = n1;
			g1List[j].y = n2;
			
			k = SegList[j].seg->armID12;
			l = SegList[j].seg->armID21;
			
			b[j].x = node1->burgX[k];
			b[j].y = node1->burgY[k];
			b[j].z = node1->burgZ[k];
			
			node1->armid[k] = 2*j+0;
			node2->armid[l] = 2*j+1;
			
			g1arms[node1->armid[k]] = j*2;
			g1arms[node2->armid[l]] = j*2+1;
			
			armid[(node1->armid[k])*2+0] = node1->myTag.index;
			armid[(node1->armid[k])*2+1] = k;
			armid[(node2->armid[l])*2+0] = node2->myTag.index;
			armid[(node2->armid[l])*2+1] = l;
			
			/* id in the GPU seg vector */
			SegList[j].seg->subindex = j;
		}
		
		// Pack reduction array
		int2 *redg1_pos = (int2*)malloc(nodeCount*sizeof(int2));
		int  *redg1_ind = (int*)malloc(2*segCount*sizeof(int));
		int ind = 0;
		for (j = 0; j < nodeCount; j++) {
			redg1_pos[j].x = ind;
			for (k = 0; k < redg1[j].size(); k++) {
				redg1_ind[ind++] = redg1[j][k];
			}
			redg1_pos[j].y = ind;
		}
		if (ind != 2*segCount) Fatal("Group1 reduction array size error!");
		
		HANDLE_ERROR(cudaMalloc(&device->g1, sizeof(int2)*segCount));
		HANDLE_ERROR(cudaMalloc(&device->g1pos, sizeof(int2)*nodeCount));
		HANDLE_ERROR(cudaMalloc(&device->g1ind, sizeof(int)*2*segCount));
		HANDLE_ERROR(cudaMalloc(&device->g1arms, sizeof(int)*armCount));
					
		HANDLE_ERROR(cudaMemcpy(device->g1, g1List, sizeof(int2)*segCount, cudaMemcpyHostToDevice));
		HANDLE_ERROR(cudaMemcpy(device->g1pos, redg1_pos, sizeof(int2)*nodeCount, cudaMemcpyHostToDevice));
		HANDLE_ERROR(cudaMemcpy(device->g1ind, redg1_ind, sizeof(int)*2*segCount, cudaMemcpyHostToDevice));
		HANDLE_ERROR(cudaMemcpy(device->g1arms, g1arms, sizeof(int)*armCount, cudaMemcpyHostToDevice));
		
		free(g1List);
		free(g1arms);
		free(redg1_pos);
		free(redg1_ind);
		
/*
 *		Memory allocation on GPU
 */		
		HANDLE_ERROR(cudaMalloc(&device->r, sizeof(double3)*nodeCount));
		HANDLE_ERROR(cudaMalloc(&device->r0, sizeof(double3)*nodeCount));
		HANDLE_ERROR(cudaMalloc(&device->n, sizeof(int)*nodeCount));
		HANDLE_ERROR(cudaMalloc(&device->b, sizeof(double3)*segCount));
		HANDLE_ERROR(cudaMalloc(&device->mob, sizeof(double)*9*nodeCount));
		HANDLE_ERROR(cudaMalloc(&device->fseg, sizeof(double3)*2*segCount));
		HANDLE_ERROR(cudaMalloc(&device->Bseg, sizeof(double)*segCount));
		HANDLE_ERROR(cudaMalloc(&device->f, sizeof(double3)*nodeCount));
		HANDLE_ERROR(cudaMalloc(&device->B, sizeof(double)*nodeCount));
		HANDLE_ERROR(cudaMalloc(&device->v, sizeof(double3)*nodeCount));
		HANDLE_ERROR(cudaMalloc(&device->v0, sizeof(double3)*nodeCount));
		HANDLE_ERROR(cudaMalloc(&device->rkf, sizeof(double3)*5*nodeCount));
		HANDLE_ERROR(cudaMalloc(&device->farms, sizeof(double3)*armCount));
		
		// Error arrays
		HANDLE_ERROR(cudaMalloc(&device->e1, sizeof(double)*blocksNodes));
		HANDLE_ERROR(cudaMalloc(&device->e2, sizeof(double)*blocksNodes));
		device->e1_ptr = thrust::device_pointer_cast(device->e1);
		device->e2_ptr = thrust::device_pointer_cast(device->e2);
		
/*
 *		Copy memory from host to device
 */
		HANDLE_ERROR(cudaMemcpy(device->r, r, sizeof(double3)*nodeCount, cudaMemcpyHostToDevice));
		HANDLE_ERROR(cudaMemcpy(device->b, b, sizeof(double3)*segCount, cudaMemcpyHostToDevice));
		HANDLE_ERROR(cudaMemcpy(device->mob, mob, sizeof(double)*9*nodeCount, cudaMemcpyHostToDevice));
		

/*
 *		Perform RKF integration on GPU. First, pack the
 *      segment pairs and send them to the GPU. Then, calculate
 *      the interaction distances to assign each interaction
 *      into a group. After this, time integrate group 0 forces.
 */		
		SendSubgroupGPU(home, device, GROUP0);
		SegSegListGPU(home, device);
		RKFIntegratorGPU(home, device, GROUP0, 0);

/*
 *		Update the different groups by moving segment pairs
 *      interactions that have been flagged.
 */		
		int *g0flag = (int*)malloc(device->nSegSeg0*sizeof(int));
		HANDLE_ERROR(cudaMemcpy(g0flag, device->g0flag, sizeof(int)*device->nSegSeg0, cudaMemcpyDeviceToHost));
		
		int cntSegSeg = 0;
		for (j = 0; j < subcyc->SegSegListG0_cnt; j++) {
			if (subcyc->SegSegListG0[j].flag == 0) continue;
			
			int subGroup = g0flag[cntSegSeg];
			cntSegSeg++;
			
			if (subGroup < 0) {
				/* Interactions that are flagged -1 are to be 
				 * ignored (e.g. very small segments) */
				subcyc->SegSegListG0[j].flag = 0;
				continue;
				
			} else if (subGroup > 0) {
				MoveInteractionGroup(subcyc, j, subGroup);
				subcyc->SegSegListG0[j].flag = 0;
			}
		}
		free(g0flag);
			
		SendSubgroupGPU(home, device, GROUP4);
		SendSubgroupGPU(home, device, GROUP3);
		SendSubgroupGPU(home, device, GROUP2);
		
/*
 *		Time integrate group 1, 2, 3 and 4 interactions (subcycle).
 */		
		real8   subTime1, subTime2, subTime3, subTime4;
        real8   totalsubDT, nextDTsub, oldDTsub, newDTsub;
        int     subGroup, cutDT;
        
        //Initialize the time for each group based on whether it has any forces in it
        if (segCount > 0) subTime1 = 0.0;
        else              subTime1 = param->realdt;
		
        if (device->nSegSeg2 > 0) subTime2 = 0.0;
        else                      subTime2 = param->realdt;
		
        if (device->nSegSeg3 > 0) subTime3 = 0.0;
        else                      subTime3 = param->realdt;
		
        if (device->nSegSeg4 > 0) subTime4 = 0.0;
        else                      subTime4 = param->realdt;
		
        //Initialize some other stuff
		if (home->cycle == 0) nextDTsub = param->realdt;
		totalsubDT = 0.0;
        subcyc->numSubCycle1 = 0;
        subcyc->numSubCycle2 = 0;
        subcyc->numSubCycle3 = 0;
        subcyc->numSubCycle4 = 0;
        int oldGroup = -1;
        int nSubcyc;
        int totSubcyc = 0;

/*
 *		Subcycle until the subcycle group times (subTimei) catch up
 *		with the global group time (realdt). Note that nodal forces 
 *		will reset to zero when subcycling is performed
 */
		while ( subTime1 < param->realdt || subTime2 < param->realdt ||
                subTime3 < param->realdt || subTime4 < param->realdt ) {
            cutDT = 0;

            //The group that is furthest behind goes first
            if        ( subTime4 <= subTime3 && subTime4 <= subTime2 && subTime4 <= subTime1 ) {
                subGroup   = GROUP4;
                nextDTsub  = param->nextDTsub4;
                totalsubDT = subTime4;
				
            } else if ( subTime3 < subTime4 && subTime3 <= subTime2 && subTime3 <= subTime1 ) {
                subGroup   = GROUP3;
                nextDTsub  = param->nextDTsub3;
                totalsubDT = subTime3;
				
            } else if ( subTime2 < subTime4 && subTime2 < subTime3 && subTime2 <= subTime1 ) {
                subGroup   = GROUP2;
                nextDTsub  = param->nextDTsub2;
                totalsubDT = subTime2;
				
            } else {
                subGroup   = GROUP1;
                nextDTsub  = param->nextDTsub;
                totalsubDT = subTime1;
            }

            //If we switched groups, reset subcycle count
            if (subGroup != oldGroup) nSubcyc = 0;
            oldGroup = subGroup;

            //Make sure we don't pass the global group in time
			if (totalsubDT + nextDTsub > param->realdt) {
				oldDTsub  = nextDTsub;
				nextDTsub = param->realdt - totalsubDT;
				newDTsub  = nextDTsub;
				cutDT     = 1;
				
				if      (subGroup == GROUP1) param->nextDTsub  = nextDTsub;
				else if (subGroup == GROUP2) param->nextDTsub2 = nextDTsub;
				else if (subGroup == GROUP3) param->nextDTsub3 = nextDTsub;
				else if (subGroup == GROUP4) param->nextDTsub4 = nextDTsub;
			}
			
            //Time integrate the chosen group for one subcycle
			RKFIntegratorGPU(home, device, subGroup, nSubcyc);
			nSubcyc++;

            //Do bookkeeping on the time step and number of subcycles
			if        (subGroup == GROUP1) {
				if (cutDT && param->realdtsub == newDTsub) param->nextDTsub = oldDTsub;
				subTime1 += param->realdtsub;
				subcyc->numSubCycle1++;
			} else if (subGroup == GROUP2) {
				if (cutDT && param->realdtsub2 == newDTsub) param->nextDTsub2 = oldDTsub;
				subTime2 += param->realdtsub2;
				subcyc->numSubCycle2++;
			} else if (subGroup == GROUP3) {
				if (cutDT && param->realdtsub3 == newDTsub) param->nextDTsub3 = oldDTsub;
				subTime3 += param->realdtsub3;
				subcyc->numSubCycle3++;
			} else if (subGroup == GROUP4) {
				if (cutDT && param->realdtsub4 == newDTsub) param->nextDTsub4 = oldDTsub;
				subTime4 += param->realdtsub4;
				subcyc->numSubCycle4++;
			}
			
			totSubcyc++;
		}
		
	
/*
 *		We are done with subcycling. Now recalculate all the
 *      forces and mobilities.
 */		
		RKFIntegratorGPU(home, device, FULL, -1);

/*
 *		Unpack arms forces back on the CPU
 */		
		double3 *farms = (double3*)malloc(sizeof(double3)*armCount);
		HANDLE_ERROR(cudaMemcpy(farms, device->farms, sizeof(double3)*armCount, cudaMemcpyDeviceToHost));
		
		for (i = 0; i < armCount; i++) {
			j = armid[2*i+0];
			k = armid[2*i+1];
			node1 = home->nodeKeys[j];
			node1->armfx[k] = farms[i].x;
			node1->armfy[k] = farms[i].y;
			node1->armfz[k] = farms[i].z;
		}
		
		free(farms);
		
/*
 *		Copy new nodal positions, forces, and velocities back to the CPU
 */	
		HANDLE_ERROR(cudaMemcpy(r, device->r, sizeof(double3)*nodeCount, cudaMemcpyDeviceToHost));
		
		double3 *f = (double3*)malloc(sizeof(double3)*nodeCount);
		double3 *v = (double3*)malloc(sizeof(double3)*nodeCount);
		HANDLE_ERROR(cudaMemcpy(f, device->f, sizeof(double3)*nodeCount, cudaMemcpyDeviceToHost));
		HANDLE_ERROR(cudaMemcpy(v, device->v, sizeof(double3)*nodeCount, cudaMemcpyDeviceToHost));
		
		nodeCount = 0;
		for (i = 0; i < home->newNodeKeyPtr; i++) {
			if ((node1 = home->nodeKeys[i]) == (Node_t *)NULL) continue;
			
			node1->x = r[nodeCount].x;
			node1->y = r[nodeCount].y;
			node1->z = r[nodeCount].z;
			
			node1->fX = f[nodeCount].x;
			node1->fY = f[nodeCount].y;
			node1->fZ = f[nodeCount].z;
			
			node1->vX = v[nodeCount].x;
			node1->vY = v[nodeCount].y;
			node1->vZ = v[nodeCount].z;
			
			node1->flags &= (~NODE_RESET_FORCES);
			
			nodeCount++;
		}
		
		
/*
 *		Check that no errors have been encountered. Errors can occur
 *      when the GPU memory becomes full or fragmented. When this
 *      happens we must stop the simulation, otherwise all subsequent
 *      results will be garbage.
 */		
		CheckErrorGPU("SubcycleIntegratorGPU");
		
/*
 *		Free memory
 */		
		HANDLE_ERROR(cudaFree(device->r));
		HANDLE_ERROR(cudaFree(device->r0));
		HANDLE_ERROR(cudaFree(device->n));
		HANDLE_ERROR(cudaFree(device->b));
		HANDLE_ERROR(cudaFree(device->mob));
		HANDLE_ERROR(cudaFree(device->fseg));
		HANDLE_ERROR(cudaFree(device->Bseg));
		HANDLE_ERROR(cudaFree(device->f));
		HANDLE_ERROR(cudaFree(device->B));
		HANDLE_ERROR(cudaFree(device->v));
		HANDLE_ERROR(cudaFree(device->v0));
		HANDLE_ERROR(cudaFree(device->rkf));
		
		HANDLE_ERROR(cudaFree(device->esig));
		
		HANDLE_ERROR(cudaFree(device->g1));
		HANDLE_ERROR(cudaFree(device->g1pos));
		HANDLE_ERROR(cudaFree(device->g1ind));
		HANDLE_ERROR(cudaFree(device->g1arms));
		
		HANDLE_ERROR(cudaFree(device->farms));
		
		//if (param->elasticinteraction) {
			HANDLE_ERROR(cudaFree(device->f0));
			HANDLE_ERROR(cudaFree(device->g0));
			HANDLE_ERROR(cudaFree(device->g0pos));
			HANDLE_ERROR(cudaFree(device->g0ind));
			HANDLE_ERROR(cudaFree(device->g0flag));
			HANDLE_ERROR(cudaFree(device->g0dist2));
			HANDLE_ERROR(cudaFree(device->g0arms_pos));
			HANDLE_ERROR(cudaFree(device->g0arms_ind));
			
			HANDLE_ERROR(cudaFree(device->f2));
			HANDLE_ERROR(cudaFree(device->g2));
			HANDLE_ERROR(cudaFree(device->g2pos));
			HANDLE_ERROR(cudaFree(device->g2ind));
			HANDLE_ERROR(cudaFree(device->g2arms_pos));
			HANDLE_ERROR(cudaFree(device->g2arms_ind));
			
			HANDLE_ERROR(cudaFree(device->f3));
			HANDLE_ERROR(cudaFree(device->g3));
			HANDLE_ERROR(cudaFree(device->g3pos));
			HANDLE_ERROR(cudaFree(device->g3ind));
			HANDLE_ERROR(cudaFree(device->g3arms_pos));
			HANDLE_ERROR(cudaFree(device->g3arms_ind));
			
			HANDLE_ERROR(cudaFree(device->f4));
			HANDLE_ERROR(cudaFree(device->g4));
			HANDLE_ERROR(cudaFree(device->g4pos));
			HANDLE_ERROR(cudaFree(device->g4ind));
			HANDLE_ERROR(cudaFree(device->g4arms_pos));
			HANDLE_ERROR(cudaFree(device->g4arms_ind));
		//}
			
		if (param->elasticinteraction) {	
			if (param->forceCutOff == 0 && param->fmEnabled) {
				HANDLE_ERROR(cudaFree(device->fmm));
			}
		}
		
		HANDLE_ERROR(cudaFree(device->e1));
		HANDLE_ERROR(cudaFree(device->e2));
		
		free(r);
		free(b);
		free(mob);
		free(f);
		free(v);
		
		HANDLE_ERROR(cudaFree(device->cc));
		free(cc);
		
		free(armid);
		for (i = 0; i < home->newNodeKeyPtr; i++) {
			if ((node1 = home->nodeKeys[i]) == (Node_t *)NULL) continue;
			free(node1->armid);
			free(node1->armInt);
		}
		
		free(device);
		
		HANDLE_ERROR(cudaEventRecord(stop, 0));
		HANDLE_ERROR(cudaEventSynchronize(stop));
		float gputime;
		HANDLE_ERROR(cudaEventElapsedTime(&gputime, start, stop));
		//printf("SubcycleIntegratorGPU time: %f ms\n", gputime);
		
		return;
}

/*------------------------------------------------------------------------
 *
 *      Function:    PairSegSegForcesGPU
 *
 *-----------------------------------------------------------------------*/
__global__ void PairSegSegForcesGPU(int pairCount, double3 *r, int *n12, int *n34, double *cc, double *b1, double *b2, double *fpair)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < pairCount) {
		
		int arm = n34[i*3+0];
		int node3 = n34[i*3+1];
		int node4 = n34[i*3+2];
		
		int node1 = n12[arm*2+0];
		int node2 = n12[arm*2+1];
		
		double3 r1, r2, r3, r4;
		double3 bs1, bs2;
		double3 f1, f2, f3, f4;
		
		bs1.x = b1[arm*3+0];
		bs1.y = b1[arm*3+1];
		bs1.z = b1[arm*3+2];
		
		bs2.x = b2[i*3+0];
		bs2.y = b2[i*3+1];
		bs2.z = b2[i*3+2];
		
		r1 = r[node1];
		r2 = r[node2];
		r3 = r[node3];
		r4 = r[node4];
		
		// Cell center
		double3 rc;
		rc.x = cc[arm*3+0];
		rc.y = cc[arm*3+1];
		rc.z = cc[arm*3+2];
		
		double3 dr1, dr2;
		double  L1s, L2s;
		dr1.x = r2.x - r1.x;
		dr1.y = r2.y - r1.y;
		dr1.z = r2.z - r1.z;
		ZImageGPU(dr1);
		L1s = dr1.x*dr1.x + dr1.y*dr1.y + dr1.z*dr1.z;
		
		dr2.x = r4.x - r3.x;
		dr2.y = r4.y - r3.y;
		dr2.z = r4.z - r3.z;
		ZImageGPU(dr2);
		L2s = dr2.x*dr2.x + dr2.y*dr2.y + dr2.z*dr2.z;
		
		if (L1s < 1.e-20 || L2s < 1.e-20) {
			
			f1.x = 0.0; f1.y = 0.0; f1.z = 0.0;
			f2.x = 0.0; f2.y = 0.0; f2.z = 0.0;
			
		} else {
		
			r2.x = r1.x + dr1.x;
			r2.y = r1.y + dr1.y;
			r2.z = r1.z + dr1.z;
			
			// Cell center
			dr1.x = r3.x - rc.x;
			dr1.y = r3.y - rc.y;
			dr1.z = r3.z - rc.z;
			ZImageGPU(dr1);
			
			r3.x = rc.x + dr1.x;
			r3.y = rc.y + dr1.y;
			r3.z = rc.z + dr1.z;
			
			dr2.x = r4.x - r3.x;
			dr2.y = r4.y - r3.y;
			dr2.z = r4.z - r3.z;
			ZImageGPU(dr2);
			
			r4.x = r3.x + dr2.x;
			r4.y = r3.y + dr2.y;
			r4.z = r3.z + dr2.z;
			
			SegSegForceIsotropicGPU(r1, r2, r3, r4, bs1, bs2, f1, f2, f3, f4);
			
		}
		
		fpair[i*6+0] = f1.x;
		fpair[i*6+1] = f1.y;
		fpair[i*6+2] = f1.z;
		fpair[i*6+3] = f2.x;
		fpair[i*6+4] = f2.y;
		fpair[i*6+5] = f2.z;
		
	}
	
	return;
}

/*------------------------------------------------------------------------
 *
 *      Function:    DevicePair_t
 *
 *-----------------------------------------------------------------------*/
typedef struct {
	int      nodeSize, pairSize, armSize;
	double3  *r_host, *r;
	double   *fpair;
	int      *n12, *n34;
	double   *cc, *b1, *b2;
} DevicePair_t;

DevicePair_t *devicePair;

/*------------------------------------------------------------------------
 *
 *      Function:    InitializeNodeForceGPU
 *
 *-----------------------------------------------------------------------*/
void InitializeNodeForceGPU(Home_t *home)
{
	devicePair = (DevicePair_t*)malloc(sizeof(DevicePair_t));
	devicePair->nodeSize = 0;
	devicePair->pairSize = 0;
	devicePair->armSize = 0;
	
	return;
}

/*------------------------------------------------------------------------
 *
 *      Function:    FinalizeNodeForceGPU
 *
 *-----------------------------------------------------------------------*/
void FinalizeNodeForceGPU(Home_t *home)
{
	if (devicePair->nodeSize > 0) {
		free(devicePair->r_host);
		HANDLE_ERROR(cudaFree(devicePair->r));
	}
	if (devicePair->pairSize > 0) {
		HANDLE_ERROR(cudaFree(devicePair->fpair));
		HANDLE_ERROR(cudaFree(devicePair->n34));
		HANDLE_ERROR(cudaFree(devicePair->b2));
	}
	if (devicePair->armSize > 0) {
		HANDLE_ERROR(cudaFree(devicePair->cc));
		HANDLE_ERROR(cudaFree(devicePair->b1));
		HANDLE_ERROR(cudaFree(devicePair->n12));
	}
	free(devicePair);
	
	return;
}

/*------------------------------------------------------------------------
 *
 *      Function:    SetOneNodeForceGPU
 *      Description: Compute nodal forces on nodes being splitted.
 *                   Note: the implementation has been optimized to 
 *                   reduce memory overheads.
 *
 *-----------------------------------------------------------------------*/
void SetOneNodeForceGPU(Home_t *home, SplitSegSeg_t *splitSegSegList)
{
		int     i, nodeSize, pairSize, armSize;
		int     segPairCnt, nodeCount;
		Node_t  *node1;
		
		segPairCnt = splitSegSegList->segPairCnt;
		nodeCount = splitSegSegList->nodeCount;
		
		// Pack the nodes
		nodeSize = devicePair->nodeSize;
		if (nodeCount > nodeSize) {
			if (nodeSize > 0) {
				free(devicePair->r_host);
				HANDLE_ERROR(cudaFree(devicePair->r));
			}
			nodeSize = 2*nodeCount;
			devicePair->nodeSize = nodeSize;
			devicePair->r_host = (double3*)malloc(sizeof(double3)*nodeSize);
			HANDLE_ERROR(cudaMalloc(&devicePair->r, sizeof(double3)*nodeSize));
		}
		
		nodeCount = 0;
		for (i = 0; i < home->newNodeKeyPtr; i++) {
			if ((node1 = home->nodeKeys[i]) == (Node_t *)NULL) continue;
			devicePair->r_host[nodeCount].x = node1->x;
			devicePair->r_host[nodeCount].y = node1->y;
			devicePair->r_host[nodeCount].z = node1->z;
			nodeCount++;
		}
		
		HANDLE_ERROR(cudaMemcpy(devicePair->r, devicePair->r_host, sizeof(double3)*nodeCount, cudaMemcpyHostToDevice));
		
		
		// Pack the interactions
		pairSize = devicePair->pairSize;
		if (segPairCnt > pairSize) {
			if (pairSize > 0) {
				HANDLE_ERROR(cudaFree(devicePair->fpair));
				HANDLE_ERROR(cudaFree(devicePair->n34));
				HANDLE_ERROR(cudaFree(devicePair->b2));
			}
			pairSize = 2*segPairCnt;
			devicePair->pairSize = pairSize;
			HANDLE_ERROR(cudaMalloc(&devicePair->fpair, sizeof(double)*6*pairSize));
			HANDLE_ERROR(cudaMalloc(&devicePair->n34, sizeof(int)*pairSize*3));
			HANDLE_ERROR(cudaMalloc(&devicePair->b2, sizeof(double)*pairSize*3));
		}
		
		int nArms = splitSegSegList->nArms;
		armSize = devicePair->armSize;
		if (nArms > armSize) {
			if (armSize > 0) {
				HANDLE_ERROR(cudaFree(devicePair->cc));
				HANDLE_ERROR(cudaFree(devicePair->b1));
				HANDLE_ERROR(cudaFree(devicePair->n12));
			}
			armSize = 2*nArms;
			devicePair->armSize = armSize;
			HANDLE_ERROR(cudaMalloc(&devicePair->cc, sizeof(double)*armSize*3));
			HANDLE_ERROR(cudaMalloc(&devicePair->b1, sizeof(double)*armSize*3));
			HANDLE_ERROR(cudaMalloc(&devicePair->n12, sizeof(int)*armSize*2));
		}
		
		HANDLE_ERROR(cudaMemcpy(devicePair->cc, splitSegSegList->cc, sizeof(double)*nArms*3, cudaMemcpyHostToDevice));
		HANDLE_ERROR(cudaMemcpy(devicePair->b1, splitSegSegList->b1, sizeof(double)*nArms*3, cudaMemcpyHostToDevice));
		HANDLE_ERROR(cudaMemcpy(devicePair->n12, splitSegSegList->n12, sizeof(int)*nArms*2, cudaMemcpyHostToDevice));
		
		HANDLE_ERROR(cudaMemcpy(devicePair->n34, splitSegSegList->n34, sizeof(int)*segPairCnt*3, cudaMemcpyHostToDevice));
		HANDLE_ERROR(cudaMemcpy(devicePair->b2, splitSegSegList->b2, sizeof(double)*segPairCnt*3, cudaMemcpyHostToDevice));
			
		
		// Compute segment / segment interactions
		int threadsPerBlock, blocksPairs;
		GetThreadsPerBlock(home->deviceProp, segPairCnt, &threadsPerBlock, &blocksPairs);
		
		PairSegSegForcesGPU<<<blocksPairs,threadsPerBlock>>>(segPairCnt, devicePair->r, devicePair->n12, devicePair->n34, devicePair->cc, devicePair->b1, devicePair->b2, devicePair->fpair);
		
		HANDLE_ERROR(cudaMemcpy(splitSegSegList->fpair, devicePair->fpair, sizeof(double)*6*segPairCnt, cudaMemcpyDeviceToHost));
		
		return;
}

/*------------------------------------------------------------------------
 *
 *      Function:    LocalSegSegForcesGPU
 *
 *-----------------------------------------------------------------------*/
__global__ void LocalSegSegForcesGPU(int pairCount, double3 *r, int4 *pair, double3 *b1, double3 *b2, double3 *cc, double *fpair)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < pairCount) {
		
		int4 nodes = pair[i];
		
		double3 r1, r2, r3, r4;
		double3 bs1, bs2;
		double3 f1, f2, f3, f4;
		
		bs1 = b1[i];
		bs2 = b2[i];
		
		r1 = r[nodes.x];
		r2 = r[nodes.y];
		r3 = r[nodes.z];
		r4 = r[nodes.w];
		
		double3 dr1, dr2;
		double  L1s, L2s;
		dr1.x = r2.x - r1.x;
		dr1.y = r2.y - r1.y;
		dr1.z = r2.z - r1.z;
		ZImageGPU(dr1);
		L1s = dr1.x*dr1.x + dr1.y*dr1.y + dr1.z*dr1.z;
		
		dr2.x = r4.x - r3.x;
		dr2.y = r4.y - r3.y;
		dr2.z = r4.z - r3.z;
		ZImageGPU(dr2);
		L2s = dr2.x*dr2.x + dr2.y*dr2.y + dr2.z*dr2.z;
		
		if (L1s < 1.e-20 || L2s < 1.e-20) {
			
			f1.x = 0.0; f1.y = 0.0; f1.z = 0.0;
			f2.x = 0.0; f2.y = 0.0; f2.z = 0.0;
			f3.x = 0.0; f3.y = 0.0; f3.z = 0.0;
			f4.x = 0.0; f4.y = 0.0; f4.z = 0.0;
			
		} else {
		
			r2.x = r1.x + dr1.x;
			r2.y = r1.y + dr1.y;
			r2.z = r1.z + dr1.z;
			
			// Cell center
			double3 rc;
			rc = cc[nodes.x];
			dr1.x = r3.x - rc.x;
			dr1.y = r3.y - rc.y;
			dr1.z = r3.z - rc.z;
			ZImageGPU(dr1);
			
			r3.x = rc.x + dr1.x;
			r3.y = rc.y + dr1.y;
			r3.z = rc.z + dr1.z;
			
			dr2.x = r4.x - r3.x;
			dr2.y = r4.y - r3.y;
			dr2.z = r4.z - r3.z;
			ZImageGPU(dr2);
			
			r4.x = r3.x + dr2.x;
			r4.y = r3.y + dr2.y;
			r4.z = r3.z + dr2.z;
			
			SegSegForceIsotropicGPU(r1, r2, r3, r4, bs1, bs2, f1, f2, f3, f4);
			
		}
		
		fpair[i*12+0] = f1.x;
		fpair[i*12+1] = f1.y;
		fpair[i*12+2] = f1.z;
		fpair[i*12+3] = f2.x;
		fpair[i*12+4] = f2.y;
		fpair[i*12+5] = f2.z;
		fpair[i*12+6] = f3.x;
		fpair[i*12+7] = f3.y;
		fpair[i*12+8] = f3.z;
		fpair[i*12+9] = f4.x;
		fpair[i*12+10] = f4.y;
		fpair[i*12+11] = f4.z;
		
	}
	
	return;
}

/*------------------------------------------------------------------------
 *
 *      Function:    LocalSegForcesGPU
 *      Description: Compute the segment pairs forces required in the
 *                   LocalSegForces calculation using the GPU device.
 *
 *-----------------------------------------------------------------------*/
void LocalSegForcesGPU(Home_t *home, int segPairListCnt, SegmentPair_t *segPairList, double *fpair)
{
		int     i, armID12, armID34;
		int     cellX, cellY, cellZ;
		double  xCenter, yCenter, zCenter;
		Node_t  *node1, *node2, *node3, *node4;
		Cell_t  *cell;
		Param_t *param;
		
		param = home->param;
		
		if (segPairListCnt == 0) {
			return;
		}
		
		// Pack nodes
		int nodeCount = 0;
		for (i = 0; i < home->newNodeKeyPtr; i++) {
			if ((node1 = home->nodeKeys[i]) == (Node_t *)NULL) continue;
			node1->subindex = nodeCount;
			nodeCount++;
		}
		
		double3 *r = (double3*)malloc(sizeof(double3)*nodeCount);
		double3 *cc = (double3*)malloc(sizeof(double3)*nodeCount);
		
		nodeCount = 0;
		for (i = 0; i < home->newNodeKeyPtr; i++) {
			if ((node1 = home->nodeKeys[i]) == (Node_t *)NULL) continue;
			r[nodeCount].x = node1->x;
			r[nodeCount].y = node1->y;
			r[nodeCount].z = node1->z;
			
			/* Find cell center */
			cell = home->cellKeys[node1->cellIdx];
			cellX = cell->xIndex;
			cellY = cell->yIndex;
			cellZ = cell->zIndex;
			FindCellCenter(param, (real8)(cellX-1), (real8)(cellY-1),
                          (real8)(cellZ-1), 2, &xCenter, &yCenter, &zCenter);
			cc[nodeCount].x = xCenter;
			cc[nodeCount].y = yCenter;
			cc[nodeCount].z = zCenter;
			
			nodeCount++;
		}
		
		Device_t *device;
		device = (Device_t*)malloc(sizeof(Device_t));
		
		HANDLE_ERROR(cudaMalloc(&device->r, sizeof(double3)*nodeCount));
		HANDLE_ERROR(cudaMalloc(&device->cc, sizeof(double3)*nodeCount));
		
		HANDLE_ERROR(cudaMemcpy(device->r, r, sizeof(double3)*nodeCount, cudaMemcpyHostToDevice));
		HANDLE_ERROR(cudaMemcpy(device->cc, cc, sizeof(double3)*nodeCount, cudaMemcpyHostToDevice));
		
		
		// Pack segment / segment interactions
		int4 *pair = (int4*)malloc(sizeof(int4)*segPairListCnt);
		double3 *b1 = (double3*)malloc(sizeof(double3)*segPairListCnt);
		double3 *b2 = (double3*)malloc(sizeof(double3)*segPairListCnt);
		
		for (i = 0; i < segPairListCnt; i++) {
			node1 = segPairList[i].seg1->node1;
			node2 = segPairList[i].seg1->node2;
			node3 = segPairList[i].seg2->node1;
			node4 = segPairList[i].seg2->node2;
			
			pair[i].x = node1->subindex;
			pair[i].y = node2->subindex;
			pair[i].z = node3->subindex;
			pair[i].w = node4->subindex;
			
			armID12 = GetArmID(home, node1, node2);
			armID34 = GetArmID(home, node3, node4);
			
			b1[i].x = node1->burgX[armID12];
			b1[i].y = node1->burgY[armID12];
			b1[i].z = node1->burgZ[armID12];
			
			b2[i].x = node3->burgX[armID34];
			b2[i].y = node3->burgY[armID34];
			b2[i].z = node3->burgZ[armID34];
		}
		
		HANDLE_ERROR(cudaMalloc(&device->pair, sizeof(int4)*segPairListCnt));
		HANDLE_ERROR(cudaMalloc(&device->b1, sizeof(double3)*segPairListCnt));
		HANDLE_ERROR(cudaMalloc(&device->b2, sizeof(double3)*segPairListCnt));
		
		HANDLE_ERROR(cudaMemcpy(device->pair, pair, sizeof(int4)*segPairListCnt, cudaMemcpyHostToDevice));
		HANDLE_ERROR(cudaMemcpy(device->b1, b1, sizeof(double3)*segPairListCnt, cudaMemcpyHostToDevice));
		HANDLE_ERROR(cudaMemcpy(device->b2, b2, sizeof(double3)*segPairListCnt, cudaMemcpyHostToDevice));
		
		
		// Compute segment / segment interactions
		int threadsPerBlock, blocksSegSegs;
		GetThreadsPerBlock(home->deviceProp, segPairListCnt, &threadsPerBlock, &blocksSegSegs);
		
		HANDLE_ERROR(cudaMalloc(&device->fpair, sizeof(double)*12*segPairListCnt));
		
		LocalSegSegForcesGPU<<<blocksSegSegs,threadsPerBlock>>>(segPairListCnt, device->r, device->pair, device->b1, device->b2, device->cc, device->fpair);
		CheckErrorGPU("LocalSegSegForcesGPU");
		
		// Copy back nodal forces
		HANDLE_ERROR(cudaMemcpy(fpair, device->fpair, sizeof(double)*12*segPairListCnt, cudaMemcpyDeviceToHost));
		
		// Free memory
		HANDLE_ERROR(cudaFree(device->r));
		HANDLE_ERROR(cudaFree(device->cc));
		HANDLE_ERROR(cudaFree(device->pair));
		HANDLE_ERROR(cudaFree(device->b1));
		HANDLE_ERROR(cudaFree(device->b2));
		HANDLE_ERROR(cudaFree(device->fpair));
		
		free(r);
		free(cc);
		free(b1);
		free(b2);
		free(pair);
		
		free(device);
		
		return;
}

/*------------------------------------------------------------------------
 *
 *      Function:    InitializeParadisGPU
 *      Description: Select the GPU device and set some parameters on it
 *
 *-----------------------------------------------------------------------*/
void InitializeParadisGPU(Home_t *home)
{
	double Lx, Ly, Lz, xc, yc, zc;
	Param_t *param;
	
	param = home->param;
	
	/* Leave if we are not using the GPU subcycling integrator */
	if (strcmp(param->timestepIntegrator, "forceBsubcycle") != 0 || 
	    strcmp(param->subInteg0Integ1, "GPU") != 0) {
		return;
	}
	
#if PARALLEL
	Fatal("GPU subcycling can only be used in serial mode!");
#endif
	if (param->useLabFrame) {
		Fatal("GPU subcycling cannot be used with useLabFrame!");
	}
	if (param->elasticinteraction && param->fmEnabled == 0) {
		Fatal("GPU subcycling cannot be used with Rijm table!");
	}
	if (param->rg1 > 0) {
		Fatal("GPU subcycling cannot be used with rg1 > 0!");
	}
	if (param->mobilityType != MOB_FCC_0) {
		Fatal("GPU subcycling can only be used with MobilityLaw_FCC_0!");
	}
	
	printf("Initializing ParaDiS GPU\n");
	
	DeviceProp_t *deviceProp;
	deviceProp = (DeviceProp_t*)malloc(sizeof(DeviceProp_t));
	SelectCudaDevice(home->deviceID, deviceProp);
	home->deviceProp = deviceProp;
			
	if (param->xBoundType == Periodic) Lx = param->Lx; else Lx = 0.0;
	if (param->yBoundType == Periodic) Ly = param->Ly; else Ly = 0.0;
	if (param->zBoundType == Periodic) Lz = param->Lz; else Lz = 0.0;
		
	xc = (param->maxSideX + param->minSideX) * 0.5;
	yc = (param->maxSideY + param->minSideY) * 0.5;
	zc = (param->maxSideZ + param->minSideZ) * 0.5;
		
	SetVariablesGPU<<<1,1>>>(param->shearModulus, param->pois, param->rc, param->TensionFactor, 
			                 param->MobEdge, param->MobScrew, Lx, Ly, Lz, xc, yc, zc, 
			                 param->rTol, param->rTolth, param->rTolrel, param->Ecore);
	
	return;
}

/*------------------------------------------------------------------------
 *
 *      Function:    GetThreadsPerBlock
 *      Description: Determine the optimal number (lowest) of threads per
 *                   block as a function of the maximum number of blocks
 *
 *-----------------------------------------------------------------------*/
void GetThreadsPerBlock(DeviceProp_t *deviceProp, int threadsSize, int *threadsPerBlock, int *blockSize)
{
	*threadsPerBlock = 64;
	
	while (*threadsPerBlock <= deviceProp->maxThreadsPerBlock) {
		*blockSize = (threadsSize + *threadsPerBlock - 1) / *threadsPerBlock;
		if (*blockSize <= deviceProp->maxBlocks) {
			break;
		}
		*threadsPerBlock *= 2;
	}
		
	return;
}

/*------------------------------------------------------------------------
 *
 *      Function:    GetCudaCores
 *
 *-----------------------------------------------------------------------*/
int GetCudaCores(cudaDeviceProp devProp)
{  
	int cores = 0;
	int mp = devProp.multiProcessorCount;
	switch (devProp.major){
		case 2: // Fermi
			if (devProp.minor == 1) cores = mp * 48;
			else cores = mp * 32;
			break;
		case 3: // Kepler
			cores = mp * 192;
			break;
		case 5: // Maxwell
			cores = mp * 128;
			break;
		case 6: // Pascal
			if (devProp.minor == 1) cores = mp * 128;
			else if (devProp.minor == 0) cores = mp * 64;
			//else printf("Unknown device type\n");
			break;
		default:
			//printf("Unknown device type\n"); 
			break;
	}
	return cores;
}

/*------------------------------------------------------------------------
 *
 *      Function:    SelectCudaDevice
 * 		Description: Select the GPU device
 *
 *-----------------------------------------------------------------------*/
void SelectCudaDevice(int deviceID, DeviceProp_t *deviceProp)
{
	int nDevices, device;
	cudaGetDeviceCount(&nDevices);
	
	printf("\nAvailable GPU device(s): %d\n", nDevices);
	if (nDevices == 0) {
		Fatal("No GPU device is available on this system");
	} else {
		for (int i = 0; i < nDevices; i++) {
			cudaDeviceProp prop;
			cudaGetDeviceProperties(&prop, i);
			printf("  GPU Device ID %d: %s\n", i, prop.name);
		}
	}
	
	if (deviceID == -1) {
		if (nDevices > 1) {
/*
 * 			Loop over the GPU devices and select
 * 			that with the maximum number of processors
 */
			int max_mp = 0;
			for (int i = 0; i < nDevices; i++) {
				cudaDeviceProp prop;
				cudaGetDeviceProperties(&prop, i);
				if (max_mp < prop.multiProcessorCount) {
					max_mp = prop.multiProcessorCount;
					device = i;
				}
			}
			cudaSetDevice(device);
		} else {
			device = 0;
		}
	} else {
/*
 * 		Select the GPU device requested with the -g option
 */
		device = deviceID;
		if (device < 0 || device >= nDevices) {
			printf("\nError: GPU device ID %d is not available on this system\n", device);
			Fatal("Please select a valid device ID in the above list or do not use -g option");
		}
		cudaSetDevice(device);
	}
	
	cudaDeviceProp prop;
	cudaGetDeviceProperties(&prop, device);
	printf("\n**************************************************\n");
	printf("GPU Device ID: %d\n", device);
	printf("  Device name: %s\n", prop.name);
	printf("  Device PCI Bus id: %d\n", prop.pciBusID);
	int cores = GetCudaCores(prop);
	if (cores == 0) {
		printf("  Number of cores: unknown\n");
	} else {
		printf("  Number of cores: %d\n", cores);
	}
	printf("  Clock rate (MHz): %f\n", 1.0*prop.clockRate/1000);
	printf("  Global memory (MB): %f\n", 1.0*prop.totalGlobalMem/1.0e6);
	printf("  Peak Memory Bandwidth (GB/s): %f\n",2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
	printf("  Maximum grid size (x,y,z): %d %d %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf("  Maximum threads per block: %d\n", prop.maxThreadsPerBlock);
	printf("**************************************************\n\n");
	
	deviceProp->deviceID = device;
	deviceProp->maxBlocks = prop.maxGridSize[0];
	deviceProp->maxThreadsPerBlock = prop.maxThreadsPerBlock;
}

#endif
